#include "hip/hip_runtime.h"
#include <stdio.h>

template<typename T, int BS>
__global__ void
block_inverse (float A[], int N, int stride)
{
  __shared__ unsigned int ipiv[BS];
  __shared__ unsigned int kb;
  __shared__ T maxval[BS], mask[BS], pivotInv;
  __shared__ T Arowk[BS], Acolk[BS];
  ipiv[threadIdx.x] = threadIdx.x;
  mask[threadIdx.x] = 1.0f;
  __syncthreads();

  unsigned int tid = threadIdx.x;

  for (int k=0; k<N; k++) {
    // First, find locate of maximum of kth column, excluding the
    // first k rows through the mask.
    maxval[tid] = mask[tid] * fabsf(A[tid*stride + k]);
    __syncthreads();
    if (threadIdx.x < 16) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+16]);
    if (threadIdx.x < 8 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+8]);
    if (threadIdx.x < 4 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+4]);
    if (threadIdx.x < 2 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+2]);
    if (threadIdx.x < 1 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+1]);
    __syncthreads();
    if ((mask[tid] * fabsf(A[tid*stride + k])) > 0.999* maxval[0]) {
      kb = tid;
      pivotInv = 1.0f/A[tid*stride + k];
    }
    __syncthreads();
    // Now kb holds pivot row and pivot the value
    
    // Swap rows
    T tmp = A[k*stride+tid];
    A[k*stride +tid] = A[kb*stride+tid];
    A[kb*stride+tid] = tmp;
    
    // Swap pivot
    if (tid == 0) {
      int itmp = ipiv[kb];
      ipiv[kb] = ipiv[k];
      ipiv[k]  = itmp;
    }
    __syncthreads();

    // Col k update
    if (tid != k)
      A[stride*tid+k] = -pivotInv*A[stride*tid+k];
    else
      A[stride*k+k] = 0.0f;
    __syncthreads();

    // Rank-1 update
    Arowk[tid] = A[stride*k   + tid];
    Acolk[tid] = A[stride*tid +   k];
    __syncthreads();
    for (int i=0; i<N; i++) 
      A[i*stride+tid] += Arowk[tid]*Acolk[i];
    __syncthreads();

    // Row k update
    if (tid != k) 
      A[k*stride+tid] *= pivotInv;
    else {
      A[k*stride+k] = pivotInv;
      mask[k] = 0.0;
    }
    __syncthreads();
  }
  // Finally, do backward pivoting
  for (int i=0; i<N; i++) {
    Arowk[tid] = A[i*stride+tid];
    __syncthreads();
    A[i*stride+ipiv[tid]] = Arowk[tid];
  }
}


template<typename T, int BS>
__device__ void
block_inverse1 (T A[BS][BS+1])
{
  __shared__ unsigned int ipiv[BS];
  __shared__ unsigned int kb;
  __shared__ T maxval[BS], mask[BS], pivotInv;
  __shared__ T Arowk[BS], Acolk[BS];
  ipiv[threadIdx.x] = threadIdx.x;
  mask[threadIdx.x] = 1.0f;
  __syncthreads();

  unsigned int tid = threadIdx.x;

  for (int k=0; k<BS; k++) {
    // First, find locate of maximum of kth column, excluding the
    // first k rows through the mask.
    maxval[tid] = mask[tid] * fabsf(A[tid][k]);
    __syncthreads();
    if (threadIdx.x < 16) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+16]);
    if (threadIdx.x < 8 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+8]);
    if (threadIdx.x < 4 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+4]);
    if (threadIdx.x < 2 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+2]);
    if (threadIdx.x < 1 ) maxval[threadIdx.x] = 
      max(maxval[threadIdx.x], maxval[threadIdx.x+1]);
    __syncthreads();
    if ((mask[tid] * fabsf(A[tid][k])) > 0.999* maxval[0]) {
      kb = tid;
      pivotInv = 1.0f/A[tid][k];
    }
    __syncthreads();
    // Now kb holds pivot row and pivot the value
    
    // Swap rows
    T tmp = A[k][tid];
    A[k][tid] = A[kb][tid];
    A[kb][tid] = tmp;
    
    // Swap pivot
    if (tid == 0) {
      int itmp = ipiv[kb];
      ipiv[kb] = ipiv[k];
      ipiv[k]  = itmp;
    }
    __syncthreads();

    // Col k update
    if (tid != k)
      A[tid][k] = -pivotInv*A[tid][k];
    else
      A[k][k] = 0.0f;
    __syncthreads();

    // Rank-1 update
    Arowk[tid] = A[k][tid];
    Acolk[tid] = A[tid][k];
    __syncthreads();
    for (int i=0; i<BS; i++) 
      A[i][tid] += Arowk[tid]*Acolk[i];
    __syncthreads();

    // Row k update
    if (tid != k) 
      A[k][tid] *= pivotInv;
    else {
      A[k][k] = pivotInv;
      mask[k] = 0.0;
    }
    __syncthreads();
  }
  // Finally, do backward pivoting
  for (int i=0; i<BS; i++) {
    Arowk[tid] = A[i][tid];
    __syncthreads();
    A[i][ipiv[tid]] = Arowk[tid];
  }
}


template<typename T, int BS>
__device__ void block_mul (float A[BS][BS+1],
			   float B[BS][BS+1],
			   float C[BS][BS+1])
{
  int tid = threadIdx.x;
  for (int row=0; row<BS; row++)
    C[row][tid] = 0.0f;
  __syncthreads();

  for (int k=0; k<BS; k++)
    for (int i=0; i<BS; i++)
      C[i][tid] += A[i][k]*B[k][tid];
}  


template<typename T, int BS>
__device__ void block_mul_add (float A[BS][BS+1],
			       float B[BS][BS+1],
			       float *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[BS];

  for (int i=0; i<BS; i++) {
    Crow[tid] = C[i*Cstride + tid];
    for (int k=0; k<BS; k++) 
      Crow[tid] += A[i][k]*B[k][tid];
    C[i*Cstride + tid] = Crow[tid];
  }
  // for (int i=0; i<N; i++)
  //   for (int k=0; k<N; k++)
  //     C[i,tid] += A(i,k)*B(k,tid);

}  

template<typename T, int BS>
__device__ void block_mul_set (float A[BS][BS+1],
			       float B[BS][BS+1],
			       float *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[BS];


  for (int i=0; i<BS; i++) {
    Crow[tid] = 0.0f;
    for (int k=0; k<BS; k++) 
      Crow[tid] += A[i][k]*B[k][tid];
    C[i*Cstride + tid] = Crow[tid];
  }


  // for (int k=0; k<BS; k++) {
  //   Crow[tid] = 0.0f;
  //   for (int i=0; i<BS; i++)
  //     Crow[tid] += A[i][k]*B[k][tid];
  //   C[k*Cstride + tid] = Crow[tid];
  // }
}  





template<typename T, int BS>
__global__ void
inverse (T A[], T work[], int N, int stride)
{
  float *Atmp = work;
  float *pivot_tmp = work+N*stride;

  __shared__ T pivot[BS][BS+1], in[BS][BS+1];
  int NB = N/BS;
  if (N%BS) NB++;
  int tid = threadIdx.x;


  for (int kb=0; kb<NB; kb++) {
    // load pivot block
    int row = kb*BS;
    for (int j=0; j<BS; j++)
      if (row+tid < N)
	pivot[j][tid] = A[(row+j)*stride + row+tid];
    
    // invert pivot
    block_inverse1<T,BS> (pivot);

    // Column scaling
    int col = kb*BS;
    for (int jb=0; jb < NB; jb++) {
      int row = jb*BS;
      if (kb != jb) {
    	for (int j=0; j<BS; j++)
    	  in[j][tid] = -A[(row+j)*stride + col + tid];
    	block_mul_set<T,BS>(in, pivot, A+row*stride+col, stride);
      }
      else {
    	for (int j=0; j<BS; j++)
    	  A[(row+j)*stride + col+tid] = 0.0f;
      }
    }	

    // Save pivot to global memory here!
    // We use it for temporary space in the rank-1 update
    for (int j=0; j<BS; j++)
      pivot_tmp[j*BS+tid] = pivot[j][tid];


    // Copy Ato Atmp
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
    	Atmp[row*stride+ib*BS+tid] =  A[row*stride+ib*BS+tid];
    
    // Rank-1 update
    for (int ib=0; ib < NB; ib++) {
      for (int i=0; i<BS; i++)
    	in[i][tid] = A[(ib*BS+i)*stride + kb*BS + tid];
      for (int jb=0; jb<NB; jb++) {
    	for (int i=0; i<BS; i++) {
    	  pivot[i][tid] = A[(kb*BS+i)*stride + jb*BS + tid];
    	  // Atmp[(ib*BS+i)*stride + (jb*BS+tid)] = 
    	  //   A[(ib*BS+i)*stride + (jb*BS+tid)];
    	}
    	block_mul_add<T,BS>(in, pivot,  Atmp+(ib*BS)*stride + jb*BS,
    			    stride);
      }
    }
    // Copy Atmp back to A
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
    	A[row*stride+ib*BS+tid] =  Atmp[row*stride+ib*BS+tid];

    // Restore pivot from global memory here!
    for (int j=0; j<BS; j++)
      pivot[j][tid] = pivot_tmp[j*BS+tid];

    // Row-scaling
    for (int ib=0; ib<NB; ib++) {
      int row = kb*BS;
      int col = ib*BS;
      if (kb != ib) {
    	for (int j=0; j<BS; j++)
    	  in[j][tid] = A[(row+j)*stride + col+tid];
    	block_mul_set<T,BS>(pivot, in, A+row*stride+col, stride);
      }
      else {
    	for (int j=0; j<BS; j++) 
    	  A[(row+j)*stride + col+tid] = pivot[j][tid];
      }
    }	
  }
}



template<typename T, int BS>
__global__ void
inverse_many (T *A_list[], T *work_list[], int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ float *A, *work;
  if (tid == 0) {
    A    = A_list[blockIdx.x];
    work = work_list[blockIdx.x];
  }
  __syncthreads();

  float *Atmp = work;
  float *pivot_tmp = work+N*stride;

  __shared__ T pivot[BS][BS+1], in[BS][BS+1];
  int NB = N/BS;
  if (N%BS) NB++;


  for (int kb=0; kb<NB; kb++) {
    // load pivot block
    int row = kb*BS;
    for (int j=0; j<BS; j++)
      if (row+tid < N)
	pivot[j][tid] = A[(row+j)*stride + row+tid];
    
    // invert pivot
    block_inverse1<T,BS> (pivot);

    // Column scaling
    int col = kb*BS;
    for (int jb=0; jb < NB; jb++) {
      int row = jb*BS;
      if (kb != jb) {
    	for (int j=0; j<BS; j++)
    	  in[j][tid] = -A[(row+j)*stride + col + tid];
    	block_mul_set<T,BS>(in, pivot, A+row*stride+col, stride);
      }
      else {
    	for (int j=0; j<BS; j++)
    	  A[(row+j)*stride + col+tid] = 0.0f;
      }
    }	

    // Save pivot to global memory here!
    // We use it for temporary space in the rank-1 update
    for (int j=0; j<BS; j++)
      pivot_tmp[j*BS+tid] = pivot[j][tid];


    // Copy Ato Atmp
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
    	Atmp[row*stride+ib*BS+tid] =  A[row*stride+ib*BS+tid];
    
    // Rank-1 update
    for (int ib=0; ib < NB; ib++) {
      for (int i=0; i<BS; i++)
    	in[i][tid] = A[(ib*BS+i)*stride + kb*BS + tid];
      for (int jb=0; jb<NB; jb++) {
    	for (int i=0; i<BS; i++) {
    	  pivot[i][tid] = A[(kb*BS+i)*stride + jb*BS + tid];
    	  // Atmp[(ib*BS+i)*stride + (jb*BS+tid)] = 
    	  //   A[(ib*BS+i)*stride + (jb*BS+tid)];
    	}
    	block_mul_add<T,BS>(in, pivot,  Atmp+(ib*BS)*stride + jb*BS,
    			    stride);
      }
    }
    // Copy Atmp back to A
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
    	A[row*stride+ib*BS+tid] =  Atmp[row*stride+ib*BS+tid];

    // Restore pivot from global memory here!
    for (int j=0; j<BS; j++)
      pivot[j][tid] = pivot_tmp[j*BS+tid];

    // Row-scaling
    for (int ib=0; ib<NB; ib++) {
      int row = kb*BS;
      int col = ib*BS;
      if (kb != ib) {
    	for (int j=0; j<BS; j++)
    	  in[j][tid] = A[(row+j)*stride + col+tid];
    	block_mul_set<T,BS>(pivot, in, A+row*stride+col, stride);
      }
      else {
    	for (int j=0; j<BS; j++) 
    	  A[(row+j)*stride + col+tid] = pivot[j][tid];
      }
    }	
  }
}





#define INVERSE_BS 32

void 
test_inverse()
{
  int N = 128;
  dim3 dimBlock(32);
  dim3 dimGrid(1);

  float *A_d, *Atmp_d, *work_d;
  int lwork = N*N + INVERSE_BS * INVERSE_BS;


  hipMalloc((void**)&A_d, N*N*sizeof(float));
  hipMalloc((void**)&work_d, lwork*sizeof(float));
  
  float A[N*N], Ainv[N*N];
  for (int i=0; i<N*N; i++)
    A[i] = drand48();
  hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
  
  inverse<float,INVERSE_BS><<<dimGrid,dimBlock>>> (A_d, work_d, N, N);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

  // Copy Ainv back to host memory
  
  hipMemcpy(Ainv, A_d, N*N*sizeof(float), hipMemcpyDeviceToHost);

  float error = 0.0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++) {
      float val = 0.0;
      for (int k=0; k<N; k++)
	val += Ainv[i*N+k]*A[k*N+j];
      float diff = (i==j) ? (1.0f-val) : val;
      error += diff*diff;
    }
  fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));

}



void 
test_inverse_many()
{
  int numMats = 1000;

  int N = 128;

  int lwork = N*N + INVERSE_BS * INVERSE_BS;
  fprintf (stderr, "lwork = %d\n", lwork);

  float **Alist, **worklist;
  float **Alist_d, **worklist_d;

  Alist    = (float**)malloc(numMats*sizeof(float*));
  worklist = (float**)malloc(numMats*sizeof(float*));
  hipMalloc((void**)&Alist_d,    numMats*sizeof(float*));
  hipMalloc((void**)&worklist_d, numMats*sizeof(float*));

  float A[N*N];
  for (int i=0; i<N*N; i++)
    A[i] = drand48();

  for (int mat=0; mat<numMats; mat++) {
    hipMalloc ((void**)&(Alist[mat]),    N*N*sizeof(float));
    hipMalloc ((void**)&(worklist[mat]), lwork*sizeof(float));
    hipMemcpy(Alist[mat], A, N*N*sizeof(float), hipMemcpyHostToDevice);
  }

  hipMemcpy(Alist_d   ,    Alist, numMats*sizeof(float*), 
	     hipMemcpyHostToDevice);
  hipMemcpy(worklist_d, worklist, numMats*sizeof(float*), 
	     hipMemcpyHostToDevice);
  
  dim3 dimBlock(32);
  dim3 dimGrid(numMats);

  clock_t start = clock();
  inverse_many<float,INVERSE_BS><<<dimGrid,dimBlock>>> 
    (Alist_d, worklist_d, N, N);
  hipDeviceSynchronize();
  clock_t end = clock();
  
  double time = (double)(end-start)/(double)CLOCKS_PER_SEC
    / (double)numMats;
  double rate = 1.0/time;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
	   rate);


  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

  // Copy Ainv back to host memory
  float Ainv[N*N];
  hipMemcpy(Ainv, Alist[0], N*N*sizeof(float), hipMemcpyDeviceToHost);

  float error = 0.0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++) {
      float val = 0.0;
      for (int k=0; k<N; k++)
	val += Ainv[i*N+k]*A[k*N+j];
      float diff = (i==j) ? (1.0f-val) : val;
      error += diff*diff;
    }
  fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));

}





#include <stdio.h>

main()
{
  test_inverse_many();

  // int N=32;
  // float A[N*N], Acopy[N*N];
  // float *A_d;
  
  // for (int i=0; i<N; i++)
  //   for (int j=0; j<N; j++)
  //     A[N*i+j] = Acopy[N*i+j] = (float) drand48();

  // hipMalloc ((void**)&A_d, N*N*sizeof(float));
  // hipMemcpy (A_d, A, N*N*sizeof(float),
  // 	      hipMemcpyHostToDevice);

  // dim3 dimBlock(N);
  // dim3 dimGrid(1);
  // block_inverse<float,32><<<dimGrid,dimBlock>>> (A_d, N, N);

  // hipDeviceSynchronize();
  // hipError_t err = hipGetLastError();
  // if (err != hipSuccess) {
  //   fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
  // 	     hipGetErrorString(err));
  //   abort();
  // }

  // hipMemcpy (A, A_d, N*N*sizeof(float),
  // 	      hipMemcpyDeviceToHost);

  // float nrm = 0.0;
  // for (int i=0; i<N; i++)
  //   for (int j=0; j<N; j++) {
  //     float val = 0.0;
  //     for (int k=0; k<N; k++)
  // 	val += A[i*N+k] * Acopy[k*N+j];
  //     float diff = (i==j) ? 1.0-val : val;
  //     nrm += diff*diff;
  //   }
  // fprintf (stderr, "Error = %1.6e\n", sqrt(nrm/(double)(N*N)));
}
