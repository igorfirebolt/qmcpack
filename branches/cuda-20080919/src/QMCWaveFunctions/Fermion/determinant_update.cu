#include "hip/hip_runtime.h"
#define DET_BLOCK_SIZE 64

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>


// The first kernel just computes AinvT * u and also stores the kth
// col of Ainv in global memory
template<typename T>
__global__ void
update_inverse_cuda1 (T *A_g[], T *Ainv_g[], T *u_g[], 
		      T *Ainv_delta_g[], T *Ainv_colk_g[], 
		      int N, int rowstride, int k)
{
  __shared__ T *A, *Ainv, *u, *Ainv_delta, *Ainv_colk;
  if (threadIdx.x==0) {
    A           = A_g[blockIdx.y];
    Ainv        = Ainv_g[blockIdx.y];
    u           = u_g[blockIdx.y];
    Ainv_delta  = Ainv_delta_g[blockIdx.y];
    Ainv_colk   = Ainv_colk_g[blockIdx.y];
  }

  __syncthreads();

  // Store the product Ainv * u in shared memory
  __shared__ T Ainv_delta_shared[DET_BLOCK_SIZE], 
    Ainv_colk_shared[DET_BLOCK_SIZE], u_shared[DET_BLOCK_SIZE],
    uold_shared[DET_BLOCK_SIZE];
  Ainv_delta_shared[threadIdx.x] = 0.0;
  int col = blockIdx.x*DET_BLOCK_SIZE + threadIdx.x;
  int numblocks = N / DET_BLOCK_SIZE;

  // If the column I need to pull from Ainv is in this thread block
  // domain, do the following
  if (blockIdx.x*DET_BLOCK_SIZE <= k && k < (blockIdx.x+1)*DET_BLOCK_SIZE) {
    for (int block=0; block<numblocks; block++) {
      u_shared[threadIdx.x] = u[block*DET_BLOCK_SIZE+threadIdx.x];
      uold_shared[threadIdx.x] = 
      	A[k*rowstride + block*DET_BLOCK_SIZE+threadIdx.x];
      // Write new row into A matrix
      A[k*rowstride + block*DET_BLOCK_SIZE+threadIdx.x] = u_shared[threadIdx.x];
      __syncthreads();
      for (int i=0; i<DET_BLOCK_SIZE; i++) {
      	int row = block*DET_BLOCK_SIZE + i;
	
      	T a = Ainv[row*rowstride+col];
      	if (col == k)
      	  Ainv_colk_shared[i] = a;
      	Ainv_delta_shared[threadIdx.x] += a*(u_shared[i]-uold_shared[i]);
      }
      __syncthreads();
      Ainv_colk[block*DET_BLOCK_SIZE+threadIdx.x] = Ainv_colk_shared[threadIdx.x];
    }
  }
  else {
    for (int block=0; block<numblocks; block++) {
      u_shared[threadIdx.x] = u[block*DET_BLOCK_SIZE+threadIdx.x];
      uold_shared[threadIdx.x] = 
  	A[k*rowstride + block*DET_BLOCK_SIZE+threadIdx.x];
      // Write new row into A matrix
      A[k*rowstride + block*DET_BLOCK_SIZE+threadIdx.x] = u_shared[threadIdx.x];
      __syncthreads();
      for (int i=0; i<DET_BLOCK_SIZE; i++) {
  	int row = block*DET_BLOCK_SIZE + i;
  	Ainv_delta_shared[threadIdx.x] += 
  	  Ainv[row*rowstride+col]*(u_shared[i]- uold_shared[i]);
      }
    }
  }

  __syncthreads();
  
  // Write the data back to global memory
  Ainv_delta[col]    = Ainv_delta_shared[threadIdx.x];
}


template<typename T>
__global__ void
update_inverse_cuda2 (T *Ainv_g[], T *u_g[], T *Ainv_delta_g[],
		      T *Ainv_colk_g[], int N, int rowstride, int k)
{
  __shared__ T *Ainv, *Ainv_delta, *Ainv_colk;
  if (threadIdx.x==0) {
    Ainv     = Ainv_g[blockIdx.y];
    Ainv_delta    = Ainv_delta_g[blockIdx.y];
    Ainv_colk = Ainv_colk_g[blockIdx.y];
  }
  __syncthreads();

  __shared__ T Ainv_delta_shared[DET_BLOCK_SIZE];
  __shared__ T  Ainv_colk_shared[DET_BLOCK_SIZE];
  int col = blockIdx.x*DET_BLOCK_SIZE + threadIdx.x;
  // Read the data back from global memory
  Ainv_delta_shared[threadIdx.x] = Ainv_delta[col];
  Ainv_colk_shared[threadIdx.x] = Ainv_colk[col];
  __shared__ T prefact;
  if (threadIdx.x == 0)
    prefact = -1.0f/(1.0f+Ainv_delta[k]);
  __syncthreads();
		   
  int numblocks = N / DET_BLOCK_SIZE;
  for (int block=0; block<numblocks; block++) {
    Ainv_colk_shared[threadIdx.x] = 
      prefact*Ainv_colk[block*DET_BLOCK_SIZE+threadIdx.x];
    __syncthreads();
    for (int i=0; i<DET_BLOCK_SIZE; i++) {
      int row = block*DET_BLOCK_SIZE + i;
      Ainv[row*rowstride+col] += 
	Ainv_delta_shared[threadIdx.x]*Ainv_colk_shared[i];
    }
  }
}


void
update_inverse_cuda(float *A_g[], float *Ainv_g[], float *u_g[], 
		    float *Ainv_delta_g[], float *Ainv_colk_g[], 
		    int N, int rowstride, int iat, int numWalkers)
{
  dim3 dimBlock(DET_BLOCK_SIZE);
  dim3 dimGrid(N/DET_BLOCK_SIZE, numWalkers);

  update_inverse_cuda1<float><<<dimGrid,dimBlock>>>
    (A_g, Ainv_g, u_g, Ainv_delta_g, Ainv_colk_g, N, rowstride, iat);
  update_inverse_cuda2<float><<<dimGrid,dimBlock>>>
    (Ainv_g, u_g, Ainv_delta_g, Ainv_colk_g, N, rowstride, iat);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in update_inverse_cuda:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}

void
update_inverse_cuda(double *A_g[], double *Ainv_g[], double *u_g[], 
		    double *Ainv_delta_g[], double *Ainv_colk_g[], 
		    int N, int rowstride, int iat, int numWalkers)
{
  dim3 dimBlock(DET_BLOCK_SIZE);
  dim3 dimGrid(N/DET_BLOCK_SIZE, numWalkers);

  fprintf (stderr, "dimBlock = %d\n", dimBlock.x);
  fprintf (stderr, "dimGrid  = (%d, %d)\n", dimGrid.x, dimGrid.y);

  update_inverse_cuda1<double><<<dimGrid,dimBlock>>>
    (A_g, Ainv_g, u_g, Ainv_delta_g, Ainv_colk_g, N, rowstride, iat);
  update_inverse_cuda2<double><<<dimGrid,dimBlock>>>
    (Ainv_g, u_g, Ainv_delta_g, Ainv_colk_g, N, rowstride, iat);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in update_inverse_cuda:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}


template<typename T, int BS>
__global__ void
calc_ratios (T *Ainv_list[], T *new_row_list[], 
	     T *ratio, int N, int row_stride, int elec)
{
  int tid = threadIdx.x;

  int col = /*blockIdx.x*BS * */tid;
  __shared__ T *Ainv, *new_row;

  if (tid == 0) {
    Ainv = Ainv_list[blockIdx.x];
    new_row = new_row_list[blockIdx.x];
  }
  __syncthreads();
  __shared__ T new_row_shared[BS];
   
  if (col < N) 
    new_row_shared[tid] = new_row[tid];
    
  __shared__ T Ainv_colk_shared[BS];
  // This is *highly* uncoallesced, but we just have to eat it to allow
  // other kernels to operate quickly.
  if (col < N)
    Ainv_colk_shared[tid] = Ainv[col*row_stride + elec];
  __syncthreads();

  __shared__ T Ainv_new_row[BS];
  if (col < N)
    Ainv_new_row[tid] = Ainv_colk_shared[tid] * new_row_shared[tid];
    
  __syncthreads();
    // Now, we have to dot
  for (unsigned int s=BS/2; s>0; s>>=1) {
    if (tid < s && (tid+s) < N)
      Ainv_new_row[tid] += Ainv_new_row[tid + s];
    __syncthreads();
  }
  if (tid == 0)      ratio[blockIdx.x] = Ainv_new_row[0];
}


void
determinant_ratios_cuda (float *Ainv_list[], float *new_row_list[],
			 float *ratios, int N, int row_stride, int iat,
			 int numWalkers)
{
  dim3 dimBlock(N);
  dim3 dimGrid(numWalkers);

  hipDeviceSynchronize();
  hipError_t err1 = hipGetLastError();
  if (err1 != hipSuccess) {
    fprintf (stderr, "CUDA error before determinant_ratios_cuda:\n  %s\n",
	     hipGetErrorString(err1));
    abort();
  }

  if (N <= 32) 
    calc_ratios<float,32><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 64)
    calc_ratios<float,64><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 128)
    calc_ratios<float,128><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 256)
    calc_ratios<float,256><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 512)
    calc_ratios<float,512><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 1024)
    calc_ratios<float,1024><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else {
    fprintf (stdout, "Error:  N too large for CUDA evaluation.\n");
    abort();
  }

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in determinant_ratios_cuda:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}

void
determinant_ratios_cuda (double *Ainv_list[], double *new_row_list[],
			 double *ratios, int N, int row_stride, int iat,
			 int numWalkers)
{
  dim3 dimBlock(N);
  dim3 dimGrid(numWalkers);

  if (N <= 32) 
    calc_ratios<double,32><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 64)
    calc_ratios<double,64><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 128)
    calc_ratios<double,128><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 256)
    calc_ratios<double,256><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else if (N <= 512)
    calc_ratios<double,512><<<dimGrid,dimBlock>>>(Ainv_list, new_row_list, ratios, N, row_stride, iat);
  else {
    fprintf (stdout, "Error:  N too large for CUDA evaluation.\n");
    abort();
  }
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in determinant_ratios_cuda:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}

