#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <complex>

using namespace std;


template<typename T, int LMAX, int BS> __global__ void
MakeAtomicOrbList (T* elec_list, int num_elecs, T* ion_list, T* radii,
		   int num_ions, T** out_list, T *L, T *Linv)
{


}

template<typename T, int LMAX, int BS> __global__ void
CalcYlmComplex (T *rhats, 
		T **Ylm_ptr, T **dYlm_dtheta_ptr, T **dYlm_dphi_ptr, int N)
{
  const T fourPiInv = 0.0795774715459477f;
  int tid = threadIdx.x;
  const int numlm = (LMAX+1)*(LMAX+1);

  __shared__ T* Ylm[BS], *dtheta[BS], *dphi[BS];
  if (blockIdx.x*BS+tid < N) {
    Ylm[tid]    = Ylm_ptr[blockIdx.x*BS+tid];
    dtheta[tid] = dYlm_dtheta_ptr[blockIdx.x*BS+tid];
    dphi[tid]   = dYlm_dphi_ptr[blockIdx.x*BS+tid];
  }

  __shared__ T rhat[BS][3];
  for (int i=0; i<3; i++) {
    int off = (3*blockIdx.x + i)*BS + tid;
    if (off < 3*N)
      rhat[0][i*BS+tid] = rhats[off];
  }
  __syncthreads();

  T costheta = rhat[tid][2];
  T sintheta = sqrt(1.0f-costheta*costheta);
  T cottheta = costheta/sintheta;
  
  T cosphi, sinphi;
  cosphi=rhat[tid][0]/sintheta;
  sinphi=rhat[tid][1]/sintheta;
  __shared__ T phi[BS];
  phi[tid] = atan2f(sinphi, cosphi);

  __shared__ T XlmVec[BS][(LMAX+1)*(LMAX+2)/2], 
    dXlmVec[BS][(LMAX+1)*(LMAX+2)/2];
  
  // Create a map from lm linear index to l and m
  __shared__ int l_lm[numlm];
  __shared__ int m_lm[numlm];
  __shared__ T  floatm_lm[numlm];
  int off=0;
  for (int l=0; l<=LMAX; l++) {
    if (tid < 2*l+1) {
      l_lm[off+tid] = l;
      m_lm[off+tid] = tid-l;
      floatm_lm[off+tid] = (T)(tid-l);
    }
    off += 2*l+1;
  }

  T lsign = 1.0f;
  T dl = 0.0f;
  for (int l=0; l<=LMAX; l++) {
    int index=l*(l+3)/2;
    XlmVec[tid][index]  = lsign;  
    dXlmVec[tid][index] = dl * cottheta * XlmVec[tid][index];
    T dm = dl;
    for (int m=l; m>0; m--, index--) {
      T tmp = sqrt((dl+dm)*(dl-dm+1.0f));
      XlmVec[tid][index-1]  = 
	-(dXlmVec[tid][index] + dm*cottheta*XlmVec[tid][index])/ tmp;
      dXlmVec[tid][index-1] = 
	(dm-1.0f)*cottheta*XlmVec[tid][index-1] + XlmVec[tid][index]*tmp;
      dm -= 1.0f;
    }
    index = l*(l+1)/2;
    T sum = XlmVec[tid][index] * XlmVec[tid][index];
    for (int m=1; m<=l; m++) 
      sum += 2.0f*XlmVec[tid][index+m]*XlmVec[tid][index+m];
    // Now, renormalize the Ylms for this l
    T norm = sqrt((2.0f*dl+1.0f)*fourPiInv / sum);
    for (int m=0; m<=l; m++) {
      XlmVec[tid][index+m]  *= norm;
      dXlmVec[tid][index+m] *= norm;
    }
    lsign *= -1.0f;
    dl += 1.0f;
  }
  __syncthreads();

  // Multiply by azimuthal phase and store in Ylm
  int end = min (N-blockIdx.x*BS, BS);
  int nb = ((LMAX+1)*(LMAX+1)+BS-1)/BS;
  __shared__ T outbuff[3][BS][2];
  for (int i=0; i < end; i++) {
    // __shared__ T sincosphi[2*LMAX+1][2];
    // if (tid < LMAX
    for (int block=0; block<nb; block++) {
      int lm = block*BS + tid;
      if (lm < numlm) {
	int l = l_lm[lm];
	int m = m_lm[lm];
	T fm = floatm_lm[lm];
	T re, im;
	__sincosf(fm*phi[i], &im, &re);
	// Switch sign if m<0 and it's odd
	if (m<0 && (m&1)) {
	  re *= -1.0f;
	  im *= -1.0f;
	}
	int off = ((l*(l+1))>>1) + abs(m);
	// Ylm
	outbuff[0][tid][0] =     re *  XlmVec[i][off];
	outbuff[0][tid][1] =     im *  XlmVec[i][off];
	// dYlm_dtheta
	outbuff[1][tid][0] =     re * dXlmVec[i][off];
	outbuff[1][tid][1] =     im * dXlmVec[i][off];
	// dYlm_dphi
	outbuff[2][tid][0] = -fm*im *  XlmVec[i][off];
	outbuff[2][tid][1] =  fm*re *  XlmVec[i][off];
      }
      __syncthreads();
      // Now write back to global mem with coallesced writes
      int off = 2*block*BS + tid;
      if (off < 2*numlm) {
	Ylm[i][off]    = outbuff[0][0][tid];
	dtheta[i][off] = outbuff[1][0][tid];
	dphi[i][off]   = outbuff[2][0][tid];
      }
      off += BS;
      if (off < 2*numlm) {
	Ylm[i][off]    = outbuff[0][0][tid+BS];
	dtheta[i][off] = outbuff[1][0][tid+BS];
	dphi[i][off]   = outbuff[2][0][tid+BS];
      }
    }
  }



  // complex<T> e2imphi (1.0, 0.0);
  // complex<T> eye(0.0, 1.0);
  // for (int m=0; m<=l; m++) {
  //   Ylm[l*(l+1)+m]  =  XlmVec[tid][l+m]*e2imphi;
  //   Ylm[l*(l+1)-m]  =  XlmVec[tid][l-m]*conj(e2imphi);
  //   dYlm_dphi[l*(l+1)+m ]  =  (double)m * eye *XlmVec[tid][l+m]*e2imphi;
  //   dYlm_dphi[l*(l+1)-m ]  = -(double)m * eye *XlmVec[tid][l-m]*conj(e2imphi);
  //   dYlm_dtheta[l*(l+1)+m] = dXlmVec[tid][l+m]*e2imphi;
  //   dYlm_dtheta[l*(l+1)-m] = dXlmVec[tid][l-m]*conj(e2imphi);
  //   e2imphi *= e2iphi;
  // } 
  
  // dl += 1.0;
  // lsign *= -1.0;
  // YlmTimer.stop();
}


template<typename T, int LMAX, int BS> __global__ void
CalcYlmReal (T *rhats, 
	     T **Ylm_ptr, T **dYlm_dtheta_ptr, T **dYlm_dphi_ptr, int N)
{
  const T fourPiInv = 0.0795774715459477f;
  int tid = threadIdx.x;
  const int numlm = (LMAX+1)*(LMAX+2)/2;

  __shared__ T* Ylm[BS], *dtheta[BS], *dphi[BS];
  if (blockIdx.x*BS+tid < N) {
    Ylm[tid]    = Ylm_ptr[blockIdx.x*BS+tid];
    dtheta[tid] = dYlm_dtheta_ptr[blockIdx.x*BS+tid];
    dphi[tid]   = dYlm_dphi_ptr[blockIdx.x*BS+tid];
  }

  __shared__ T rhat[BS][3];
  for (int i=0; i<3; i++) {
    int off = (3*blockIdx.x + i)*BS + tid;
    if (off < 3*N)
      rhat[0][i*BS+tid] = rhats[off];
  }
  __syncthreads();

  T costheta = rhat[tid][2];
  T sintheta = sqrt(1.0f-costheta*costheta);
  T cottheta = costheta/sintheta;
  
  T cosphi, sinphi;
  cosphi=rhat[tid][0]/sintheta;
  sinphi=rhat[tid][1]/sintheta;
  __shared__ T phi[BS];
  phi[tid] = atan2f(sinphi, cosphi);

  __shared__ T XlmVec[BS][numlm], 
    dXlmVec[BS][numlm];
  
  // Create a map from lm linear index to l and m
  __shared__ int l_lm[numlm];
  __shared__ int m_lm[numlm];
  __shared__ T  floatm_lm[numlm];
  int off=0;
  for (int l=0; l<=LMAX; l++) {
    if (tid < l+1) {
      l_lm[off+tid] = l;
      m_lm[off+tid] = tid;
      floatm_lm[off+tid] = (T)tid;
    }
    off += l+1;
  }

  T lsign = 1.0f;
  T dl = 0.0f;
  for (int l=0; l<=LMAX; l++) {
    int index=l*(l+3)/2;
    XlmVec[tid][index]  = lsign;  
    dXlmVec[tid][index] = dl * cottheta * XlmVec[tid][index];
    T dm = dl;
    for (int m=l; m>0; m--, index--) {
      T tmp = sqrt((dl+dm)*(dl-dm+1.0f));
      XlmVec[tid][index-1]  = 
	-(dXlmVec[tid][index] + dm*cottheta*XlmVec[tid][index])/ tmp;
      dXlmVec[tid][index-1] = 
	(dm-1.0f)*cottheta*XlmVec[tid][index-1] + XlmVec[tid][index]*tmp;
      dm -= 1.0f;
    }
    index = l*(l+1)/2;
    T sum = XlmVec[tid][index] * XlmVec[tid][index];
    for (int m=1; m<=l; m++) 
      sum += 2.0f*XlmVec[tid][index+m]*XlmVec[tid][index+m];
    // Now, renormalize the Ylms for this l
    T norm = sqrt((2.0f*dl+1.0f)*fourPiInv / sum);
    for (int m=0; m<=l; m++) {
      XlmVec[tid][index+m]  *= norm;
      dXlmVec[tid][index+m] *= norm;
    }
    lsign *= -1.0f;
    dl += 1.0f;
  }
  __syncthreads();

  // Multiply by azimuthal phase and store in Ylm
  int end = min (N-blockIdx.x*BS, BS);
  int nb = (numlm+BS-1)/BS;
  __shared__ T outbuff[3][2*BS];
  for (int i=0; i < end; i++) {
    for (int block=0; block<nb; block++) {
      int lm = block*BS + tid;
      if (lm < numlm) {
	int l = l_lm[lm];
	int m = m_lm[lm];
	T fm = floatm_lm[lm];
	T re, im;
	__sincosf(fm*phi[i], &im, &re);
	int off = ((l*(l+1))>>1) + m;
	int iplus = l*(l+1)+m;
	int iminus = l*(l+1)-m;
	// Ylm
	outbuff[0][iplus] =     re *  XlmVec[i][off];
	// dYlm_dtheta
	outbuff[1][iplus] =     re * dXlmVec[i][off];
	// dYlm_dphi
	outbuff[2][iplus] = -fm*im *  XlmVec[i][off];
	if (m != 0) {
	  outbuff[0][iminus] =     im *  XlmVec[i][off];
	  outbuff[1][iminus] =     im * dXlmVec[i][off];
	  outbuff[2][iminus] =  fm*re *  XlmVec[i][off];
	}
      }
      __syncthreads();
      // Now write back to global mem with coallesced writes
      int off = block*BS + tid;
      if (off < (LMAX+1)*(LMAX+1)) {
	Ylm[i][off]    = outbuff[0][tid];
	dtheta[i][off] = outbuff[1][tid];
	dphi[i][off]   = outbuff[2][tid];
      }
      off += BS;
      if (off < (LMAX+1)*(LMAX+1)) {
	Ylm[i][off]    = outbuff[0][tid+BS];
	dtheta[i][off] = outbuff[1][tid+BS];
	dphi[i][off]   = outbuff[2][tid+BS];
      }
    }
  }
}

template<typename T>
void CalcYlmRealCuda (T *rhats, 
		      T **Ylm_ptr, T **dYlm_dtheta_ptr, T **dYlm_dphi_ptr, 
		      int lMax, int N)
{
  const int BS=32;
  int Nblocks = (N+BS-1)/BS;
  dim3 dimGrid(Nblocks);
  dim3 dimBlock(BS);
  
  if (lMax == 0)
    return;
  else if (lMax == 1)
    CalcYlmReal<T,1,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 2)
    CalcYlmReal<T,2,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 3)
    CalcYlmReal<T,3,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 4)
    CalcYlmReal<T,4,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 5)
    CalcYlmReal<T,5,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 6)
    CalcYlmReal<T,6,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 7)
    CalcYlmReal<T,7,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 8)
    CalcYlmReal<T,8,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 9)
    CalcYlmReal<T,9,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);
  else if (lMax == 10)
    CalcYlmReal<T,10,BS><<<dimGrid,dimBlock>>>(rhats,Ylm_ptr,dYlm_dtheta_ptr,N);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in CalcYlmRealCuda:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


#ifdef TEST_GPU_YLM

class Vec3
{
private:
  double r[3];
public:
  inline double  operator[](int i) const { return r[i]; }
  inline double& operator[](int i) { return r[i];}
  Vec3(double x, double y, double z) 
  { r[0]=x; r[1]=y; r[2]=z; }
  Vec3() { }
};
  

// Fast implementation
// See Geophys. J. Int. (1998) 135,pp.307-309
void
CalcYlm (Vec3 rhat,
	 vector<complex<double> > &Ylm,
	 vector<complex<double> > &dYlm_dtheta,
	 vector<complex<double> > &dYlm_dphi,
	 int lMax)
{
  const double fourPiInv = 0.0795774715459477;
  
  double costheta = rhat[2];
  double sintheta = std::sqrt(1.0-costheta*costheta);
  double cottheta = costheta/sintheta;
  
  double cosphi, sinphi;
  cosphi=rhat[0]/sintheta;
  sinphi=rhat[1]/sintheta;
  
  complex<double> e2iphi(cosphi, sinphi);
  
  
  double lsign = 1.0;
  double dl = 0.0;
  double XlmVec[2*lMax+1], dXlmVec[2*lMax+1];
  for (int l=0; l<=lMax; l++) {
    XlmVec[2*l]  = lsign;  
    dXlmVec[2*l] = dl * cottheta * XlmVec[2*l];
    XlmVec[0]    = lsign*XlmVec[2*l];
    dXlmVec[0]   = lsign*dXlmVec[2*l];
    double dm = dl;
    double msign = lsign;
    for (int m=l; m>0; m--) {
      double tmp = std::sqrt((dl+dm)*(dl-dm+1.0));
      XlmVec[l+m-1]  = -(dXlmVec[l+m] + dm*cottheta*XlmVec[l+m])/ tmp;
      dXlmVec[l+m-1] = (dm-1.0)*cottheta*XlmVec[l+m-1] + XlmVec[l+m]*tmp;
      // Copy to negative m
      XlmVec[l-(m-1)]  = -msign* XlmVec[l+m-1];
      dXlmVec[l-(m-1)] = -msign*dXlmVec[l+m-1];
      msign *= -1.0;
      dm -= 1.0;
    }
    double sum = 0.0;
    for (int m=-l; m<=l; m++) 
      sum += XlmVec[l+m]*XlmVec[l+m];
    // Now, renormalize the Ylms for this l
    double norm = std::sqrt((2.0*dl+1.0)*fourPiInv / sum);
    for (int m=-l; m<=l; m++) {
      XlmVec[l+m]  *= norm;
      dXlmVec[l+m] *= norm;
    }
    
    // Multiply by azimuthal phase and store in Ylm
    complex<double> e2imphi (1.0, 0.0);
    complex<double> eye(0.0, 1.0);
    for (int m=0; m<=l; m++) {
      Ylm[l*(l+1)+m]  =  XlmVec[l+m]*e2imphi;
      Ylm[l*(l+1)-m]  =  XlmVec[l-m]*conj(e2imphi);
      dYlm_dphi[l*(l+1)+m ]  =  (double)m * eye *XlmVec[l+m]*e2imphi;
      dYlm_dphi[l*(l+1)-m ]  = -(double)m * eye *XlmVec[l-m]*conj(e2imphi);
      dYlm_dtheta[l*(l+1)+m] = dXlmVec[l+m]*e2imphi;
      dYlm_dtheta[l*(l+1)-m] = dXlmVec[l-m]*conj(e2imphi);
      e2imphi *= e2iphi;
    } 
    
    dl += 1.0;
    lsign *= -1.0;
  }
}

// Fast implementation
// See Geophys. J. Int. (1998) 135,pp.307-309
void
CalcYlm (Vec3 rhat,
	 vector<double> &Ylm,
	 vector<double> &dYlm_dtheta,
	 vector<double> &dYlm_dphi,
	 int lMax)
{
  const double fourPiInv = 0.0795774715459477;
    
  double costheta = rhat[2];
  double sintheta = std::sqrt(1.0-costheta*costheta);
  double cottheta = costheta/sintheta;
    
  double cosphi, sinphi;
  cosphi=rhat[0]/sintheta;
  sinphi=rhat[1]/sintheta;
    
  complex<double> e2iphi(cosphi, sinphi);
    
  double lsign = 1.0;
  double dl = 0.0;
  double XlmVec[2*lMax+1], dXlmVec[2*lMax+1];
  for (int l=0; l<=lMax; l++) {
    XlmVec[2*l]  = lsign;  
    dXlmVec[2*l] = dl * cottheta * XlmVec[2*l];
    XlmVec[0]    = lsign*XlmVec[2*l];
    dXlmVec[0]   = lsign*dXlmVec[2*l];
    double dm = dl;
    double msign = lsign;
    for (int m=l; m>0; m--) {
      double tmp = std::sqrt((dl+dm)*(dl-dm+1.0));
      XlmVec[l+m-1]  = -(dXlmVec[l+m] + dm*cottheta*XlmVec[l+m])/ tmp;
      dXlmVec[l+m-1] = (dm-1.0)*cottheta*XlmVec[l+m-1] + XlmVec[l+m]*tmp;
      // Copy to negative m
      XlmVec[l-(m-1)]  = -msign* XlmVec[l+m-1];
      dXlmVec[l-(m-1)] = -msign*dXlmVec[l+m-1];
      msign *= -1.0;
      dm -= 1.0;
    }
    double sum = 0.0;
    for (int m=-l; m<=l; m++) 
      sum += XlmVec[l+m]*XlmVec[l+m];
    // Now, renormalize the Ylms for this l
    double norm = std::sqrt((2.0*dl+1.0)*fourPiInv / sum);
    for (int m=-l; m<=l; m++) {
      XlmVec[l+m]  *= norm;
      dXlmVec[l+m] *= norm;
    }
      
    // Multiply by azimuthal phase and store in Ylm
    Ylm[l*(l+1)]         =  XlmVec[l];
    dYlm_dphi[l*(l+1) ]  = 0.0;
    dYlm_dtheta[l*(l+1)] = dXlmVec[l];
    complex<double> e2imphi = e2iphi;
    for (int m=1; m<=l; m++) {
      Ylm[l*(l+1)+m]         =  XlmVec[l+m]*e2imphi.real();
      Ylm[l*(l+1)-m]         =  XlmVec[l+m]*e2imphi.imag();
      dYlm_dphi[l*(l+1)+m ]  = -(double)m * XlmVec[l+m] *e2imphi.imag();
      dYlm_dphi[l*(l+1)-m ]  =  (double)m * XlmVec[l+m] *e2imphi.real();
      dYlm_dtheta[l*(l+1)+m] = dXlmVec[l+m]*e2imphi.real();
      dYlm_dtheta[l*(l+1)-m] = dXlmVec[l+m]*e2imphi.imag();
      e2imphi *= e2iphi;
    } 
      
    dl += 1.0;
    lsign *= -1.0;
  }
}




#include <stdlib.h>

void TestYlmComplex()
{
  int numr = 1000;
  const int BS=32;

  float *rhat_device, *Ylm_device, *dtheta_device, *dphi_device;
  float **Ylm_ptr, **dtheta_ptr, **dphi_ptr;
  const int lmax = 5;
  const int numlm = (lmax+1)*(lmax+1);

  hipMalloc ((void**)&rhat_device, 3*sizeof(float)*numr);
  hipMalloc ((void**)&Ylm_device, 2*numlm*sizeof(float)*numr);
  hipMalloc ((void**)&dtheta_device, 2*numlm*sizeof(float)*numr);
  hipMalloc ((void**)&dphi_device, 2*numlm*sizeof(float)*numr);
  hipMalloc ((void**)&Ylm_ptr,    numr*sizeof(float*));
  hipMalloc ((void**)&dtheta_ptr, numr*sizeof(float*));
  hipMalloc ((void**)&dphi_ptr,   numr*sizeof(float*));
  
  float *Ylm_host[numr], *dtheta_host[numr], *dphi_host[numr];
  float rhost[3*numr];
  vector<Vec3> rlist;
  for (int i=0; i<numr; i++) {
    Vec3 r;
    r[0] = 2.0*drand48()-1.0;
    r[1] = 2.0*drand48()-1.0;
    r[2] = 2.0*drand48()-1.0;
    double nrm = 1.0/std::sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    r[0] *= nrm;        r[1] *= nrm;        r[2] *= nrm;
    rlist.push_back(r);
    rhost[3*i+0]=r[0];  rhost[3*i+1]=r[1];  rhost[3*i+2]=r[2];
    
    Ylm_host[i] = Ylm_device+2*i*numlm;    
    dtheta_host[i] = dtheta_device+2*i*numlm;
    dphi_host[i]   = dphi_device + 2*i*numlm;
  }
  
  hipMemcpy(rhat_device, rhost, 3*numr*sizeof(float),  hipMemcpyHostToDevice);
  hipMemcpy(Ylm_ptr, Ylm_host, numr*sizeof(float*),    hipMemcpyHostToDevice);
  hipMemcpy(dtheta_ptr, dtheta_host, numr*sizeof(float*), 
	     hipMemcpyHostToDevice);
  hipMemcpy(dphi_ptr,  dphi_host, numr*sizeof(float*), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in hipMemcpy:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }


  dim3 dimBlock(BS);
  dim3 dimGrid((numr+BS-1)/BS);
  
  clock_t start, end;

  start = clock();
  for (int i=0; i<10000; i++) {
    CalcYlmComplex<float,5,BS><<<dimGrid,dimBlock>>>
      (rhat_device, Ylm_ptr, dtheta_ptr, dphi_ptr, numr);
  }
  hipDeviceSynchronize();
  end = clock();
  fprintf (stderr, "Ylm rate = %1.8f\n",
	   10000*numr/((double)(end-start)/(double)CLOCKS_PER_SEC));

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in CalcYlmComplex:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
  complex<float> Ylm[numr*numlm], dtheta[numr*numlm], dphi[numr*numlm];
  hipMemcpy(Ylm, Ylm_device, 2*numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipMemcpy(dtheta, dtheta_device, 2*numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipMemcpy(dphi, dphi_device, 2*numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in hipMemcpy:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

  int n = 999;
  vector<complex<double> > Ylm_cpu(numlm), dtheta_cpu(numlm), dphi_cpu(numlm);
  CalcYlm (rlist[n], Ylm_cpu, dtheta_cpu, dphi_cpu, lmax);
  fprintf (stderr, "Ylm:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.7f %12.7f   %12.7f %12.7f  %3.0f %3.0f\n",
	    Ylm_cpu[lm].real(), Ylm_cpu[lm].imag(), 
	    Ylm[lm+n*numlm].real(), Ylm[lm+n*numlm].imag(),
	    Ylm_cpu[lm].real()/Ylm[lm+n*numlm].real(),
	    Ylm_cpu[lm].imag()/Ylm[lm+n*numlm].imag());
  }

  fprintf (stderr, "dtheta:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.6f %12.6f   %12.6f %12.6f  %3.0f %3.0f\n",
	    dtheta_cpu[lm].real(), dtheta_cpu[lm].imag(), 
	    dtheta[lm+n*numlm].real(), dtheta[lm+n*numlm].imag(),
	    dtheta_cpu[lm].real()/dtheta[lm+n*numlm].real(),
	    dtheta_cpu[lm].imag()/dtheta[lm+n*numlm].imag());
  }

  fprintf (stderr, "dphi:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.6f %12.6f   %12.6f %12.6f  %3.0f %3.0f\n",
	    dphi_cpu[lm].real(), dphi_cpu[lm].imag(), 
	    dphi[lm+n*numlm].real(), dphi[lm+n*numlm].imag(),
	    dphi_cpu[lm].real()/dphi[lm+n*numlm].real(),
	    dphi_cpu[lm].imag()/dphi[lm+n*numlm].imag());
  }
}


void TestYlmReal()
{
  int numr = 1000;
  const int BS=32;

  float *rhat_device, *Ylm_device, *dtheta_device, *dphi_device;
  float **Ylm_ptr, **dtheta_ptr, **dphi_ptr;
  const int lmax = 5;
  const int numlm = (lmax+1)*(lmax+1);

  hipMalloc ((void**)&rhat_device,   3*sizeof(float)*numr);
  hipMalloc ((void**)&Ylm_device,    numlm*sizeof(float)*numr);
  hipMalloc ((void**)&dtheta_device, numlm*sizeof(float)*numr);
  hipMalloc ((void**)&dphi_device,   numlm*sizeof(float)*numr);
  hipMalloc ((void**)&Ylm_ptr,       numr*sizeof(float*));
  hipMalloc ((void**)&dtheta_ptr,    numr*sizeof(float*));
  hipMalloc ((void**)&dphi_ptr,      numr*sizeof(float*));
  
  float *Ylm_host[numr], *dtheta_host[numr], *dphi_host[numr];
  float rhost[3*numr];
  vector<Vec3> rlist;
  for (int i=0; i<numr; i++) {
    Vec3 r;
    r[0] = 2.0*drand48()-1.0;
    r[1] = 2.0*drand48()-1.0;
    r[2] = 2.0*drand48()-1.0;
    double nrm = 1.0/std::sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    r[0] *= nrm;        r[1] *= nrm;        r[2] *= nrm;
    rlist.push_back(r);
    rhost[3*i+0]=r[0];  rhost[3*i+1]=r[1];  rhost[3*i+2]=r[2];
    
    Ylm_host[i]    = Ylm_device    + i*numlm;    
    dtheta_host[i] = dtheta_device + i*numlm;
    dphi_host[i]   = dphi_device   + i*numlm;
  }
  
  hipMemcpy(rhat_device, rhost, 3*numr*sizeof(float),  hipMemcpyHostToDevice);
  hipMemcpy(Ylm_ptr, Ylm_host, numr*sizeof(float*),    hipMemcpyHostToDevice);
  hipMemcpy(dtheta_ptr, dtheta_host, numr*sizeof(float*), 
	     hipMemcpyHostToDevice);
  hipMemcpy(dphi_ptr,  dphi_host, numr*sizeof(float*), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in hipMemcpy:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }


  dim3 dimBlock(BS);
  dim3 dimGrid((numr+BS-1)/BS);
  
  clock_t start, end;

  start = clock();
  for (int i=0; i<10000; i++) {
    CalcYlmReal<float,lmax,BS><<<dimGrid,dimBlock>>>
      (rhat_device, Ylm_ptr, dtheta_ptr, dphi_ptr, numr);
  }
  hipDeviceSynchronize();
  end = clock();
  fprintf (stderr, "Ylm rate = %1.8f\n",
	   10000*numr/((double)(end-start)/(double)CLOCKS_PER_SEC));

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in CalcYlmReal:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
  float Ylm[numr*numlm], dtheta[numr*numlm], dphi[numr*numlm];
  hipMemcpy(Ylm, Ylm_device, numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipMemcpy(dtheta, dtheta_device, numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipMemcpy(dphi, dphi_device, numr*numlm*sizeof(float), 
	     hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in hipMemcpy:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

  int n = 999;
  vector<double> Ylm_cpu(numlm), dtheta_cpu(numlm), dphi_cpu(numlm);
  CalcYlm (rlist[n], Ylm_cpu, dtheta_cpu, dphi_cpu, lmax);
  fprintf (stderr, "Ylm:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.7f %12.7f %3.0f\n",
	    Ylm_cpu[lm], 
	    Ylm[lm+n*numlm], 
	    Ylm_cpu[lm]/Ylm[lm+n*numlm]);
  }

  fprintf (stderr, "dtheta:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.6f %12.6f %3.0f \n",
	    dtheta_cpu[lm], 
	    dtheta[lm+n*numlm], 
	    dtheta_cpu[lm]/dtheta[lm+n*numlm]);
  }

  fprintf (stderr, "dphi:\n");
  for (int lm=0; lm<numlm; lm++) {
    fprintf(stderr, "%12.6f %12.6f %3.0f\n",
	    dphi_cpu[lm], 
	    dphi[lm+n*numlm], 
	    dphi_cpu[lm]/dphi[lm+n*numlm]);
  }
}




main()
{
  TestYlmComplex();
  TestYlmReal();
}
#endif
