#include "hip/hip_runtime.h"
#define MAX_SPLINES 100
#include <stdio.h>
#include "BsplineJastrowCuda.h"

bool AisInitialized = false;


// void
// createCudaSplines (float rmax, int N,
// 		   float f[], float df[], float d2f[],
// 		   int &fSpline, int &dfSpline, int &d2fSpline)
// {
//   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
//   hipArray *fArray, *dfArray, *d2fArray;
//   hipMallocArray(  &fArray, &channelDesc, N);
//   hipMallocArray( &dfArray, &channelDesc, N);
//   hipMallocArray(&d2fArray, &channelDesc, N);
  
//   hipMemcpyToArray(fArray,  N,1,  f,N*sizeof(float),hipMemcpyHostToDevice);
//   hipMemcpyToArray(dfArray, N,1, df,N*sizeof(float),hipMemcpyHostToDevice);
//   hipMemcpyToArray(d2fArray,N,1,d2f,N*sizeof(float),hipMemcpyHostToDevice);


//   hipBindTextureToArray(texSplines[fSpline=curTex++], fArray);
//   hipBindTextureToArray(texSplines[dfSpline=curTex++], dfArray);
//   hipBindTextureToArray(texSplines[d2fSpline=curTex++], d2fArray);
// }


template<typename T>
__device__
T min_dist (T& x, T& y, T& z, 
	    T L[3][3], T Linv[3][3])
{
  T u0 = Linv[0][0]*x + Linv[1][0]*y + Linv[2][0]*z;  
  T u1 = Linv[0][1]*x + Linv[1][1]*y + Linv[2][1]*z;
  T u2 = Linv[0][2]*x + Linv[1][2]*y + Linv[2][2]*z;

  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);

  x = L[0][0]*u0 + L[1][0]*u1 + L[2][0]*u2;
  y = L[0][1]*u0 + L[1][1]*u1 + L[2][1]*u2;
  z = L[0][2]*u0 + L[1][2]*u1 + L[2][2]*u2;

//   T u0 = Linv[0][0]*x; u0 -= rintf(u0); x = L[0][0]*u0;
//   T u1 = Linv[1][1]*y; u1 -= rintf(u1); y = L[1][1]*u1;
//   T u2 = Linv[2][2]*z; u2 -= rintf(u2); z = L[2][2]*u2;
//   return sqrtf(x*x + y*y + z*z);

  T d2min = x*x + y*y + z*z;
  for (T i=-1.0f; i<=1.001; i+=1.0f)
    for (T j=-1.0f; j<=1.001; j+=1.0f)
      for (T k=-1.0f; k<=1.001; k+=1.0f) {
	T xnew = L[0][0]*(u0+i) + L[1][0]*(u1+j) + L[2][0]*(u2+k);
	T ynew = L[0][1]*(u0+i) + L[1][1]*(u1+j) + L[2][1]*(u2+k);
	T znew = L[0][2]*(u0+i) + L[1][2]*(u1+j) + L[2][2]*(u2+k);
	
	T d2 = xnew*xnew + ynew*ynew + znew*znew;
	//	d2min = min (d2, d2min);
	if (d2 < d2min) {
	  d2min = d2;
	  x = xnew;
	  y = ynew;
	  z = znew;
	}
      }
  return sqrt(d2min);
}


template<typename T>
__device__
T min_dist_fast (T& x, T& y, T& z, 
		 T L[3][3], T Linv[3][3])
{
  T u0 = Linv[0][0]*x + Linv[1][0]*y + Linv[2][0]*z;  
  T u1 = Linv[0][1]*x + Linv[1][1]*y + Linv[2][1]*z;
  T u2 = Linv[0][2]*x + Linv[1][2]*y + Linv[2][2]*z;

  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);

  x = L[0][0]*u0 + L[1][0]*u1 + L[2][0]*u2;
  y = L[0][1]*u0 + L[1][1]*u1 + L[2][1]*u2;
  z = L[0][2]*u0 + L[1][2]*u1 + L[2][2]*u2;

  return sqrtf(x*x + y*y + z*z);
}




template<typename T>
__device__
T min_dist (T& x, T& y, T& z, 
	    T L[3][3], T Linv[3][3],
	    T images[27][3])
{
  T u0 = Linv[0][0]*x + Linv[1][0]*y + Linv[2][0]*z;  
  T u1 = Linv[0][1]*x + Linv[1][1]*y + Linv[2][1]*z;
  T u2 = Linv[0][2]*x + Linv[1][2]*y + Linv[2][2]*z;

  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);

  T xtmp = L[0][0]*u0 + L[1][0]*u1 + L[2][0]*u2;
  T ytmp = L[0][1]*u0 + L[1][1]*u1 + L[2][1]*u2;
  T ztmp = L[0][2]*u0 + L[1][2]*u1 + L[2][2]*u2;
  x = xtmp;
  y = ytmp;
  z = ztmp;

//   T u0 = Linv[0][0]*x; u0 -= rintf(u0); x = L[0][0]*u0;
//   T u1 = Linv[1][1]*y; u1 -= rintf(u1); y = L[1][1]*u1;
//   T u2 = Linv[2][2]*z; u2 -= rintf(u2); z = L[2][2]*u2;
//   return sqrtf(x*x + y*y + z*z);

  T d2min = xtmp*xtmp + ytmp*ytmp + ztmp*ztmp;

  for (int i=0; i<27; i++) {
    T xnew = xtmp + images[i][0];
    T ynew = ytmp + images[i][1];
    T znew = ztmp + images[i][2];
    T d2 = xnew*xnew + ynew*ynew + znew*znew;
    if (d2 < d2min) {
      x = xnew;
      y = ynew;
      z = znew;
      d2min = d2;
    }
    __syncthreads();
  }
  return sqrt(d2min);
}


template<typename T>
__device__
T min_dist_only (T x, T y, T z, 
		 T L[3][3], T Linv[3][3],
		 T images[27][3])
{
  T u0 = Linv[0][0]*x + Linv[1][0]*y + Linv[2][0]*z;  
  T u1 = Linv[0][1]*x + Linv[1][1]*y + Linv[2][1]*z;
  T u2 = Linv[0][2]*x + Linv[1][2]*y + Linv[2][2]*z;

  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);

  x = L[0][0]*u0 + L[1][0]*u1 + L[2][0]*u2;
  y = L[0][1]*u0 + L[1][1]*u1 + L[2][1]*u2;
  z = L[0][2]*u0 + L[1][2]*u1 + L[2][2]*u2;

  T d2min = x*x + y*y + z*z;

  for (int i=0; i<27; i++) {
    T xnew = x + images[i][0];
    T ynew = y + images[i][1];
    T znew = z + images[i][2];
    T d2 = xnew*xnew + ynew*ynew + znew*znew;
    d2min = min (d2min, d2);
    __syncthreads();
  }
  return sqrt(d2min);
}



__constant__ float AcudaSpline[48];
__constant__ double AcudaSpline_double[48];

void
cuda_spline_init()
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline), A_h, 48*sizeof(float), 0, 
		     hipMemcpyHostToDevice);

  double A_d[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline_double), A_d, 48*sizeof(double), 0, 
		     hipMemcpyHostToDevice);

  AisInitialized = true;
}


template<typename T>
__device__ T 
eval_1d_spline(T dist, T rmax, T drInv, T A[4][4], T coefs[])
{
  if (dist >= rmax)  return (T)0.0;

  T s = dist * drInv;
  T sf = floorf (s);
  int index = (int)sf;
  T t = s - sf;
  T t2 = t*t;
  T t3 = t*t2;

//   return (coefs[index+0]*(AcudaSpline[ 0]*t*t*t + AcudaSpline[ 1]*t*t + AcudaSpline[ 2]*t + AcudaSpline[ 3]) +
//   	  coefs[index+1]*(AcudaSpline[ 4]*t*t*t + AcudaSpline[ 5]*t*t + AcudaSpline[ 6]*t + AcudaSpline[ 7]) +
//   	  coefs[index+2]*(AcudaSpline[ 8]*t*t*t + AcudaSpline[ 9]*t*t + AcudaSpline[10]*t + AcudaSpline[11]) +
//   	  coefs[index+3]*(AcudaSpline[12]*t*t*t + AcudaSpline[13]*t*t + AcudaSpline[14]*t + AcudaSpline[15]));


  return (coefs[index+0]*(A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]) +
  	  coefs[index+1]*(A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]) +
  	  coefs[index+2]*(A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]) +
  	  coefs[index+3]*(A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]));
}


template<typename T>
__device__ void 
eval_1d_spline_vgl(T dist, T rmax, T drInv, T A[12][4], T coefs[],
		   T& u, T& du, T& d2u)
{
  if (dist >= rmax) {
    u = du = d2u = (T)0.0;
    return;
  }

  T s = dist * drInv;
  T sf = floorf (s);
  int index = (int)sf;
  T t = s - sf;
  T t2 = t*t;
  T t3 = t*t2;

  u = (coefs[index+0]*(A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]) +
       coefs[index+1]*(A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]) +
       coefs[index+2]*(A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]) +
       coefs[index+3]*(A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]));

 du = drInv *    
   (coefs[index+0]*(A[4][0]*t3 + A[4][1]*t2 + A[4][2]*t + A[4][3]) +
    coefs[index+1]*(A[5][0]*t3 + A[5][1]*t2 + A[5][2]*t + A[5][3]) +
    coefs[index+2]*(A[6][0]*t3 + A[6][1]*t2 + A[6][2]*t + A[6][3]) +
    coefs[index+3]*(A[7][0]*t3 + A[7][1]*t2 + A[7][2]*t + A[7][3]));
 
 d2u = drInv*drInv * 
   (coefs[index+0]*(A[ 8][0]*t3 + A[ 8][1]*t2 + A[ 8][2]*t + A[ 8][3]) +
    coefs[index+1]*(A[ 9][0]*t3 + A[ 9][1]*t2 + A[ 9][2]*t + A[ 9][3]) +
    coefs[index+2]*(A[10][0]*t3 + A[10][1]*t2 + A[10][2]*t + A[10][3]) +
    coefs[index+3]*(A[11][0]*t3 + A[11][1]*t2 + A[11][2]*t + A[11][3]));
}



#define MAX_COEFS 32
template<typename T, int BS >
__global__ void
two_body_sum_kernel(T **R, int e1_first, int e1_last, 
		    int e2_first, int e2_last,
		    T *spline_coefs, int numCoefs, T rMax,  
		    T *lattice, T* latticeInv, T* sum)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  


  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[4][4];
  if (tid < 16)
    A[tid>>2][tid&3] = AcudaSpline[tid];
  __syncthreads();


  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  T mysum = (T)0.0; 
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) 
  	r1[0][i*BS + tid] = myR[3*e1_first + (3*b1+i)*BS + tid];
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
  	if (ptcl1 != ptcl2 && (ptcl1 < (N1+e1_first) ) && (ptcl2 < (N2+e2_first)))
	  mysum += eval_1d_spline (dist, rMax, drInv, A, coefs);
      }
      __syncthreads();
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s=BS>>1; s>0; s >>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  T factor = (e1_first == e2_first) ? 0.5 : 1.0;

  if (tid==0)
    sum[blockIdx.x] += factor*shared_sum[0];

}

void
two_body_sum (float *R[], int e1_first, int e1_last, int e2_first, int e2_last,
	      float spline_coefs[], int numCoefs, float rMax,  
	      float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_sum_kernel<float,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_sum:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}


void
two_body_sum (double *R[], int e1_first, int e1_last, int e2_first, int e2_last,
	      double spline_coefs[], int numCoefs, double rMax,  
	      double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_sum_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_sum:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}




template<typename T, int BS>
__global__ void
two_body_ratio_kernel(T **R, int first, int last,
		      T *Rnew, int inew,
		      T *spline_coefs, int numCoefs, T rMax,  
		      T *lattice, T* latticeInv, T* sum)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  


  int tid = threadIdx.x;
  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();

  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);

  __shared__ T shared_sum[BS];
  shared_sum[tid] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][n] = myR[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    T dx, dy, dz;
    dx = myRnew[0] - r1[tid][0];
    dy = myRnew[1] - r1[tid][1];
    dz = myRnew[2] - r1[tid][2];
    T dist = min_dist(dx, dy, dz, L, Linv);
    T delta = eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv);
    delta -= eval_1d_spline (dist, rMax, drInv, A, coefs);
    
    if (ptcl1 != inew && (ptcl1 < (N+first) ))
      shared_sum[tid] += delta;
    __syncthreads();
  }
  __syncthreads();
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  if (tid==0)
    sum[blockIdx.x] += shared_sum[0];
}




void
two_body_ratio (float *R[], int first, int last,
		float Rnew[], int inew,
		float spline_coefs[], int numCoefs, float rMax,  
		float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_ratio_kernel<float,BS><<<dimGrid,dimBlock>>>
    (R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_ratio1:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



void
two_body_ratio (double *R[], int first, int last,
		double Rnew[], int inew,
		double spline_coefs[], int numCoefs, double rMax,  
		double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  dim3 dimBlock(128);
  dim3 dimGrid(numWalkers);

  two_body_ratio_kernel<double,128><<<dimGrid,dimBlock>>>
    (R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_ratio2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



template<typename T, int BS>
__global__ void
two_body_ratio_grad_kernel(T **R, int first, int last,
			   T *Rnew, int inew,
			   T *spline_coefs, int numCoefs, T rMax,  
			   T *lattice, T* latticeInv, 
			   bool zero, T *ratio_grad)
{
  int tid = threadIdx.x;
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  int index=0;
  __shared__ T images[27][3];
  if (tid < 3)
    for (T i=-1.0; i<=1.001; i+=1.0)
      for (T j=-1.0; j<=1.001; j+=1.0)
	for (T k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);

  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid] = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][n] = myR[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    T dx, dy, dz, u, du, d2u, delta, dist;
    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv, images);
    delta = -eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRnew[0] - r1[tid][0];
    dy = myRnew[1] - r1[tid][1];
    dz = myRnew[2] - r1[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv, images);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs,
			u, du, d2u);
    delta += u;
    
    if (ptcl1 != inew && (ptcl1 < (N+first) )) {
      du /= dist;
      shared_sum[tid] += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  __syncthreads();
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
      shared_grad[tid][0] += shared_grad[tid+s][0];
      shared_grad[tid][1] += shared_grad[tid+s][1];
      shared_grad[tid][2] += shared_grad[tid+s][2];
    __syncthreads();
  }

  if (tid==0) {
    if (zero) {
      ratio_grad[4*blockIdx.x+0] = shared_sum[0];
      ratio_grad[4*blockIdx.x+1] = shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] = shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] = shared_grad[0][2];
    }
    else {
      ratio_grad[4*blockIdx.x+0] += shared_sum[0];
      ratio_grad[4*blockIdx.x+1] += shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] += shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] += shared_grad[0][2];
    }
  }
}



template<typename T, int BS>
__global__ void
two_body_ratio_grad_kernel_fast (T **R, int first, int last,
				 T *Rnew, int inew,
				 T *spline_coefs, int numCoefs, T rMax,  
				 T *lattice, T* latticeInv, 
				 bool zero, T* ratio_grad)
{
  int tid = threadIdx.x;
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  


  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int N = last - first + 1;
  int NB = (N+BS-1)/BS;

  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid] = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][n] = myR[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    T dx, dy, dz, u, du, d2u, delta, dist;
    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    dist = min_dist_fast(dx, dy, dz, L, Linv);
    delta = -eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRnew[0] - r1[tid][0];
    dy = myRnew[1] - r1[tid][1];
    dz = myRnew[2] - r1[tid][2];
    dist = min_dist_fast(dx, dy, dz, L, Linv);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs,
			u, du, d2u);
    delta += u;
    
    if (ptcl1 != inew && (ptcl1 < (N+first) )) {
      du /= dist;
      shared_sum[tid] += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  __syncthreads();
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
      shared_grad[tid][0] += shared_grad[tid+s][0];
      shared_grad[tid][1] += shared_grad[tid+s][1];
      shared_grad[tid][2] += shared_grad[tid+s][2];
    __syncthreads();
  }

  if (tid==0) {
    if (zero) {
      ratio_grad[4*blockIdx.x+0] = shared_sum[0];
      ratio_grad[4*blockIdx.x+1] = shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] = shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] = shared_grad[0][2];
    }
    else {
      ratio_grad[4*blockIdx.x+0] += shared_sum[0];
      ratio_grad[4*blockIdx.x+1] += shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] += shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] += shared_grad[0][2];
    }
  }
}




// use_fast_image indicates that Rmax < simulation cell radius.  In
// this case, we don't have to search over 27 images.
void
two_body_ratio_grad(float *R[], int first, int last,
		    float  Rnew[], int inew,
		    float spline_coefs[], int numCoefs, float rMax,  
		    float lattice[], float latticeInv[], bool zero,
		    float ratio_grad[], int numWalkers,
		    bool use_fast_image)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  // fprintf(stderr, "first = %d\n", first);
  // fprintf(stderr, "last  = %d\n", last);
  // fprintf(stderr, "inew  = %d\n", inew);
  // fprintf(stderr, "rMax = %1.3f\n", rMax);
  if (use_fast_image) 
    two_body_ratio_grad_kernel_fast<float,BS><<<dimGrid,dimBlock>>>
      (R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax,
       lattice, latticeInv, zero, ratio_grad);
  else
    two_body_ratio_grad_kernel<float,BS><<<dimGrid,dimBlock>>>
      (R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax,
       lattice, latticeInv, zero, ratio_grad);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_ratio_grad:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}


void
two_body_ratio_grad(double *R[], int first, int last,
		    double  Rnew[], int inew,
		    double spline_coefs[], int numCoefs, double rMax,  
		    double lattice[], double latticeInv[], bool zero,
		    double ratio_grad[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  
  two_body_ratio_grad_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax,
     lattice, latticeInv, zero, ratio_grad);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_ratio_grad_2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}

  


template<int BS>
__global__ void
two_body_NLratio_kernel(NLjobGPU<float> *jobs, int first, int last,
			float** spline_coefs, int *numCoefs, float *rMaxList, 
			float* lattice, float* latticeInv, 
			float sim_cell_radius)
{
  const int MAX_RATIOS = 18;
  int tid = threadIdx.x;
  __shared__ NLjobGPU<float> myJob;
  __shared__ float myRnew[MAX_RATIOS][3], myRold[3];
  __shared__ float* myCoefs;
  __shared__ int myNumCoefs;
  __shared__ float rMax;
  if (tid == 0) {
    myJob = jobs[blockIdx.x];
    myCoefs = spline_coefs[blockIdx.x];
    myNumCoefs = numCoefs[blockIdx.x];
    rMax = rMaxList[blockIdx.x];
  }
  __syncthreads();
  bool use_fast = sim_cell_radius >= rMax;

  if (tid < 3 ) 
    myRold[tid] = myJob.R[3*myJob.Elec+tid];
  for (int i=0; i<3; i++) 
    if (i*BS + tid < 3*myJob.NumQuadPoints)
      myRnew[0][i*BS+tid] = myJob.QuadPoints[i*BS+tid];
  __syncthreads();

  float dr = rMax/(float)(myNumCoefs-3);
  float drInv = 1.0/dr;
  
  __shared__ float coefs[MAX_COEFS];
  __shared__ float r1[BS][3];
  __shared__ float L[3][3], Linv[3][3];
  
  if (tid < myNumCoefs)
    coefs[tid] = myCoefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  int index=0;
  __shared__ float images[27][3];
  if (tid < 3)
    for (float i=-1.0; i<=1.001; i+=1.0)
      for (float j=-1.0; j<=1.001; j+=1.0)
	for (float k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();
  
  __shared__ float A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();
  
  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  
  __shared__ float shared_sum[MAX_RATIOS][BS+1];
  for (int iq=0; iq<myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (float)0.0;

  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][n] = myJob.R[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    float dx, dy, dz;
    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    float dist;
    if (use_fast)
      dist = min_dist_fast(dx, dy, dz, L, Linv);
    else
      dist = min_dist_only(dx, dy, dz, L, Linv, images);

    float uOld = eval_1d_spline (dist, rMax, drInv, A, coefs);

    if (use_fast)
      for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
	dx = myRnew[iq][0] - r1[tid][0];
	dy = myRnew[iq][1] - r1[tid][1];
	dz = myRnew[iq][2] - r1[tid][2];
	dist = min_dist_fast(dx, dy, dz, L, Linv);
	if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	  shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
      }
    else
      for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
	dx = myRnew[iq][0] - r1[tid][0];
	dy = myRnew[iq][1] - r1[tid][1];
	dz = myRnew[iq][2] - r1[tid][2];
	dist = min_dist_only(dx, dy, dz, L, Linv, images);
	if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	  shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
    }

    __syncthreads();
  }
  
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) 
      for (int iq=0; iq < myJob.NumQuadPoints; iq++)
	shared_sum[iq][tid] += shared_sum[iq][tid+s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}






template<int BS>
__global__ void
two_body_NLratio_kernel(NLjobGPU<double> *jobs, int first, int last,
			double **spline_coefs, int *numCoefs, 
			double *rMaxList, 
			double *lattice, double *latticeInv, 
			double sim_cell_radius)
{
  const int MAX_RATIOS = 18;
  int tid = threadIdx.x;
  __shared__ NLjobGPU<double> myJob;
  __shared__ double myRnew[MAX_RATIOS][3], myRold[3];
  __shared__ double* myCoefs;
  __shared__ int myNumCoefs;
  __shared__ double rMax;
  if (tid == 0) {
    myJob = jobs[blockIdx.x];
    myCoefs = spline_coefs[blockIdx.x];
    myNumCoefs = numCoefs[blockIdx.x];
    rMax = rMaxList[blockIdx.x];
  }
  __syncthreads();

  if (tid < 3 ) 
    myRold[tid] = myJob.R[3*myJob.Elec+tid];
  for (int i=0; i<3; i++) 
    if (i*BS + tid < 3*myJob.NumQuadPoints)
      myRnew[0][i*BS+tid] = myJob.QuadPoints[i*BS+tid];
  __syncthreads();

  double dr = rMax/(double)(myNumCoefs-3);
  double drInv = 1.0/dr;
  

  __shared__ double coefs[MAX_COEFS];
  __shared__ double r1[BS][3];
  __shared__ double L[3][3], Linv[3][3];
  
  if (tid < myNumCoefs)
    coefs[tid] = myCoefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ double A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();
  
  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  
  __shared__ double shared_sum[MAX_RATIOS][BS+1];
  for (int iq=0; iq<myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (double)0.0;

  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][n] = myJob.R[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    double dx, dy, dz;
    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    double dist = min_dist(dx, dy, dz, L, Linv);
    double uOld = eval_1d_spline (dist, rMax, drInv, A, coefs);

    for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
      dx = myRnew[iq][0] - r1[tid][0];
      dy = myRnew[iq][1] - r1[tid][1];
      dz = myRnew[iq][2] - r1[tid][2];
      dist = min_dist(dx, dy, dz, L, Linv);
      if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) 
      for (int iq=0; iq < myJob.NumQuadPoints; iq++)
	shared_sum[iq][tid] += shared_sum[iq][tid+s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}




void
two_body_NLratios(NLjobGPU<float> jobs[], int first, int last,
		  float* spline_coefs[], int numCoefs[], float rMax[], 
		  float lattice[], float latticeInv[], float sim_cell_radius,
		  int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS=32;

  dim3 dimBlock(BS);

  while (numjobs > 65535) {
    dim3 dimGrid(65535);
    two_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
      (jobs, first, last, spline_coefs, numCoefs, rMax,
       lattice, latticeInv, sim_cell_radius);
    jobs += 65535;
    numjobs -= 65535;
  }
  dim3 dimGrid(numjobs);
  two_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
    (jobs, first, last, spline_coefs, numCoefs, rMax,
     lattice, latticeInv, sim_cell_radius);
  
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_NLratios:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
  
}


void
two_body_NLratios(NLjobGPU<double> jobs[], int first, int last,
		  double* spline_coefs[], int numCoefs[], double rMax[], 
		  double lattice[], double latticeInv[], 
		  double sim_cell_radius, int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;

  dim3 dimBlock(BS);

  while (numjobs > 65535) {
    dim3 dimGrid(65535);
    two_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
      (jobs, first, last, spline_coefs, numCoefs, rMax,
       lattice, latticeInv, sim_cell_radius);
    jobs += 65535;
    numjobs -= 65535;
  }
  dim3 dimGrid(numjobs);
  two_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
    (jobs, first, last, spline_coefs, numCoefs, rMax,
     lattice, latticeInv, sim_cell_radius);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_NLratios2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}



template<typename T>
__global__ void
two_body_update_kernel (T **R, int N, int iat)
{
  __shared__ T* myR;
  if (threadIdx.x == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  
  if (threadIdx.x < 3)
    myR[3*iat + threadIdx.x] = myR[3*N + threadIdx.x];
}


void
two_body_update(float *R[], int N, int iat, int numWalkers)
{
  dim3 dimBlock(32);
  dim3 dimGrid(numWalkers);

  two_body_update_kernel<float><<<dimGrid, dimBlock>>> (R, N, iat);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_update:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}

void
two_body_update(double *R[], int N, int iat, int numWalkers)
{
  dim3 dimBlock(3);
  dim3 dimGrid(numWalkers);

  two_body_update_kernel<double><<<dimGrid, dimBlock>>> (R, N, iat);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_update2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}





#define MAX_COEFS 32

template<typename T, int BS>
__global__ void
two_body_grad_lapl_kernel(T **R, int e1_first, int e1_last, 
			  int e2_first, int e2_last,
			  T *spline_coefs, int numCoefs, T rMax,  
			  T *lattice, T *latticeInv, 
			  T *gradLapl, int row_stride)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;
  
  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  __shared__ T sGradLapl[BS][4];
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) 
  	r1[0][i*BS + tid] = myR[3*e1_first + (3*b1+i)*BS + tid];
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    int offset = blockIdx.x * row_stride + 4*b1*BS + 4*e1_first;
    sGradLapl[tid][0] = sGradLapl[tid][1] = 
      sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz, u, du, d2u;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
	eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
  	if (ptcl1 != ptcl2 && (ptcl1 < (N1+e1_first) ) && (ptcl2 < (N2+e2_first))) {
	  du /= dist;
	  sGradLapl[tid][0] += du * dx;
	  sGradLapl[tid][1] += du * dy;
	  sGradLapl[tid][2] += du * dz;
	  sGradLapl[tid][3] -= d2u + 2.0*du;
	}
      }
      __syncthreads();
    }
    for (int i=0; i<4; i++)
      if ((4*b1+i)*BS + tid < 4*N1)
	gradLapl[offset + i*BS +tid] += sGradLapl[0][i*BS+tid];
    __syncthreads();
  }
}


template<typename T, int BS>
__global__ void
two_body_grad_lapl_kernel_fast(T **R, int e1_first, int e1_last, 
			       int e2_first, int e2_last,
			       T *spline_coefs, int numCoefs, T rMax,  
			       T *lattice, T *latticeInv, 
			       T *gradLapl, int row_stride)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;
  

  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  __shared__ T sGradLapl[BS][4];
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) 
  	r1[0][i*BS + tid] = myR[3*e1_first + (3*b1+i)*BS + tid];
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    int offset = blockIdx.x * row_stride + 4*b1*BS + 4*e1_first;
    sGradLapl[tid][0] = sGradLapl[tid][1] = 
      sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz, u, du, d2u;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
	eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
  	if (ptcl1 != ptcl2 && (ptcl1 < (N1+e1_first) ) && (ptcl2 < (N2+e2_first))) {
	  du /= dist;
	  sGradLapl[tid][0] += du * dx;
	  sGradLapl[tid][1] += du * dy;
	  sGradLapl[tid][2] += du * dz;
	  sGradLapl[tid][3] -= d2u + 2.0*du;
	}
      }
      __syncthreads();
    }
    for (int i=0; i<4; i++)
      if ((4*b1+i)*BS + tid < 4*N1)
	gradLapl[offset + i*BS +tid] += sGradLapl[0][i*BS+tid];
    __syncthreads();
  }
}



void
two_body_grad_lapl(float *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   float spline_coefs[], int numCoefs, float rMax,  
		   float lattice[], float latticeInv[], float sim_cell_radius,
		   float gradLapl[], int row_stride, int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  // if (sim_cell_radius >= rMax) 
  // two_body_grad_lapl_kernel_fast<float,BS><<<dimGrid,dimBlock>>>
  //   (R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
  //    rMax, lattice, latticeInv,  gradLapl, row_stride);
  // else
    two_body_grad_lapl_kernel<float,BS><<<dimGrid,dimBlock>>>
      (R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
       rMax, lattice, latticeInv,  gradLapl, row_stride);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_grad_lapl:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
two_body_grad_lapl(double *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   double spline_coefs[], int numCoefs, double rMax,  
		   double lattice[], double latticeInv[], 
		   double gradLapl[], int row_stride, int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_grad_lapl_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
     rMax, lattice, latticeInv,  gradLapl, row_stride);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_grad_lapl2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



template<typename T, int BS>
__global__ void
two_body_grad_kernel(T **R, int first, int last, int iat,
		     T *spline_coefs, int numCoefs, T rMax,  
		     T *lattice, T *latticeInv, bool zeroOut, T *grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;
  
  int tid = threadIdx.x;
  __shared__ T *myR, r2[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r2[tid] = myR[3*iat+tid];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int index=0;
  __shared__ T images[27][3];
  if (tid < 3)
    for (T i=-1.0; i<=1.001; i+=1.0)
      for (T j=-1.0; j<=1.001; j+=1.0)
	for (T k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();



  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0]   = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][i*BS + tid] = myR[3*first + (3*b+i)*BS + tid];
    __syncthreads();
    int ptcl1 = first+b*BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx = r2[0] - r1[tid][0];
    dy = r2[1] - r1[tid][1];
    dz = r2[2] - r1[tid][2];
    T dist = min_dist(dx, dy, dz, L, Linv, images);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 != iat && ptcl1 < (N+first)) {
      du /= dist;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s=BS>>1; s>0; s>>=1) {
    if (tid < s) {
      sGrad[tid][0] += sGrad[tid+s][0];
      sGrad[tid][1] += sGrad[tid+s][1];
      sGrad[tid][2] += sGrad[tid+s][2];
    }
    __syncthreads();
  }
  if (tid < 3) {
    if (zeroOut) 
      grad[3*blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3*blockIdx.x + tid] += sGrad[0][tid];
  }
}


template<typename T, int BS>
__global__ void
two_body_grad_kernel_fast(T **R, int first, int last, int iat,
			  T *spline_coefs, int numCoefs, T rMax,  
			  T *lattice, T *latticeInv, bool zeroOut, T *grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;

  
  int tid = threadIdx.x;
  __shared__ T *myR, r2[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r2[tid] = myR[3*iat+tid];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0]   = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][i*BS + tid] = myR[3*first + (3*b+i)*BS + tid];
    __syncthreads();
    int ptcl1 = first+b*BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx = r2[0] - r1[tid][0];
    dy = r2[1] - r1[tid][1];
    dz = r2[2] - r1[tid][2];
    T dist = min_dist_fast(dx, dy, dz, L, Linv);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 != iat && ptcl1 < (N+first)) {
      du /= dist;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s=BS>>1; s>0; s>>=1) {
    if (tid < s) {
      sGrad[tid][0] += sGrad[tid+s][0];
      sGrad[tid][1] += sGrad[tid+s][1];
      sGrad[tid][2] += sGrad[tid+s][2];
    }
    __syncthreads();
  }
  if (tid < 3) {
    if (zeroOut) 
      grad[3*blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3*blockIdx.x + tid] += sGrad[0][tid];
  }
}





void
two_body_gradient (float *R[], int first, int last, int iat, 
		   float spline_coefs[], int numCoefs, float rMax,
		   float lattice[], float latticeInv[], float sim_cell_radius,
		   bool zeroOut,
		   float grad[], int numWalkers)
{
  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  if (sim_cell_radius >= rMax) 
  two_body_grad_kernel_fast<float,BS><<<dimGrid,dimBlock>>>
    (R, first, last, iat, spline_coefs, numCoefs,
     rMax, lattice, latticeInv,  zeroOut, grad);
  else
    two_body_grad_kernel<float,BS><<<dimGrid,dimBlock>>>
      (R, first, last, iat, spline_coefs, numCoefs,
       rMax, lattice, latticeInv,  zeroOut, grad);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_gradient:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
two_body_gradient (double *R[], int first, int last, int iat, 
		   double spline_coefs[], int numCoefs, double rMax,
		   double lattice[], double latticeInv[], bool zeroOut,
		   double grad[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_grad_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, first, last, iat, spline_coefs, numCoefs,
     rMax, lattice, latticeInv,  zeroOut, grad);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_gradient2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}




template<typename T, int BS>
__global__ void
two_body_derivs_kernel(T **R, T **gradLogPsi,
		       int e1_first, int e1_last, 
		       int e2_first, int e2_last,
		       int numCoefs, T rMax,  
		       T *lattice, T *latticeInv, 
		       T **derivs)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0f/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;

  
  int tid = threadIdx.x;
  __shared__ T *myR, *myGrad, *myDerivs;
  if (tid == 0) {
    myR      =          R[blockIdx.x];
    myGrad   = gradLogPsi[blockIdx.x];
    myDerivs =     derivs[blockIdx.x];
  }
    
  __shared__ T sderivs[MAX_COEFS][2];
  // __shared__ T coefs[MAX_COEFS];
  // if (tid < numCoefs)
  //   coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  sderivs[tid][0] = T();
  sderivs[tid][1] = T();

  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  __shared__ T sGrad[BS][3];
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) {
	int outoff = i*BS+tid;
	int inoff  = outoff + 3*e1_first + 3*b1*BS;
  	r1[0][outoff]    =    myR[inoff];//[3*e1_first + (3*b1+i)*BS + tid];
	sGrad[0][outoff] = myGrad[inoff];
      }
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
	T distInv = 1.0f/dist;

	T s = dist * drInv;
	T sf = floorf (s);
	int index = (int)sf;
	T t = s - sf;
	T t2 = t*t;
	T t3 = t*t2;
	T v0, v1, v2, v3;
	
	    // sderivs[index+0][0] += (A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]);
	    // sderivs[index+1][0] += (A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]);
	    // sderivs[index+2][0] += (A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]);
	    // sderivs[index+3][0] += (A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]);
	v0 = (A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]);
	v1 = (A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]);
	v2 = (A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]);
	v3 = (A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]);
	for (int id=0; id<BS; id++) 
	  if (tid == id && ptcl1 != ptcl2 && ptcl1 <= e1_last && (dist < rMax)) {
	    sderivs[index+0][0] += v0;
	    sderivs[index+1][0] += v1;
	    sderivs[index+2][0] += v2;
	    sderivs[index+3][0] += v3;
	  }
	  
      	T prefact = (dx*sGrad[tid][0] + dy*sGrad[tid][1] + dz*sGrad[tid][2])*distInv;
      	T du0 = drInv * (A[4][0]*t3 + A[4][1]*t2 + A[4][2]*t + A[4][3]);
      	T du1 = drInv * (A[5][0]*t3 + A[5][1]*t2 + A[5][2]*t + A[5][3]);
      	T du2 = drInv * (A[6][0]*t3 + A[6][1]*t2 + A[6][2]*t + A[6][3]);
      	T du3 = drInv * (A[7][0]*t3 + A[7][1]*t2 + A[7][2]*t + A[7][3]);
      	// This is the dot (gradu, grad_log_psi) term.
      	v0 = 2.0f* prefact * du0;
      	v1 = 2.0f* prefact * du1;
      	v2 = 2.0f* prefact * du2;
      	v3 = 2.0f* prefact * du3;
      	// This is the lapl u term
      	v0 -= drInv*drInv*(A[ 8][0]*t3 + A[ 8][1]*t2 + A[ 8][2]*t + A[ 8][3]) + 2.0f*du0*distInv;
      	v1 -= drInv*drInv*(A[ 9][0]*t3 + A[ 9][1]*t2 + A[ 9][2]*t + A[ 9][3]) + 2.0f*du1*distInv;
      	v2 -= drInv*drInv*(A[10][0]*t3 + A[10][1]*t2 + A[10][2]*t + A[10][3]) + 2.0f*du2*distInv;
      	v3 -= drInv*drInv*(A[11][0]*t3 + A[11][1]*t2 + A[11][2]*t + A[11][3]) + 2.0f*du3*distInv;

	for (int id=0; id<BS; id++) 
	  if (tid == id && ptcl1 != ptcl2 && ptcl1 <= e1_last && (dist < rMax)) {
	    sderivs[index+0][1] += v0;
	    sderivs[index+1][1] += v1;
	    sderivs[index+2][1] += v2;
	    sderivs[index+3][1] += v3;
	  }
      }
	  
      __syncthreads();
    }
  }
  //  if (e1_first == e2_first)
  sderivs[tid][0] *= 0.5f;
  sderivs[tid][1] *= 0.5f;
  
  if (tid < 2*numCoefs) 
    myDerivs[tid] = -sderivs[0][tid];
  if (tid+BS < 2*numCoefs)
    myDerivs[tid+BS] = sderivs[0][tid+BS];
  
}


void
two_body_derivs(float *R[], float *gradLogPsi[], int e1_first, int e1_last, 
		int e2_first, int e2_last,
		int numCoefs, float rMax,  
		float lattice[], float latticeInv[], float sim_cell_radius,
		float *derivs[], int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  if (sim_cell_radius >= rMax) 
    two_body_derivs_kernel<float,BS><<<dimGrid,dimBlock>>>
      (R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, 
       rMax, lattice, latticeInv, derivs);
  else
    two_body_derivs_kernel<float,BS><<<dimGrid,dimBlock>>>
      (R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, 
       rMax, lattice, latticeInv, derivs);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_derivs:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}

void
two_body_derivs(double *R[], double *gradLogPsi[], int e1_first, int e1_last, 
		int e2_first, int e2_last,
		int numCoefs, double rMax,  
		double lattice[], double latticeInv[], double sim_cell_radius,
		double *derivs[], int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  if (sim_cell_radius >= rMax) 
    two_body_derivs_kernel<double,BS><<<dimGrid,dimBlock>>>
      (R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, 
       rMax, lattice, latticeInv, derivs);
  else
    two_body_derivs_kernel<double,BS><<<dimGrid,dimBlock>>>
      (R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, 
       rMax, lattice, latticeInv, derivs);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in two_body_derivs:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



////////////////////////////////////////////////////////////////
//                      One-body routines                     //
////////////////////////////////////////////////////////////////

template<typename T, int BS >
__global__ void
one_body_sum_kernel(T *C, T **R, int cfirst, int clast, 
		    int efirst, int elast,
		    T *spline_coefs, int numCoefs, T rMax,  
		    T *lattice, T *latticeInv, T *sum)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T rc[BS][3], re[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[4][4];
  if (tid < 16)
    A[tid>>2][tid&3] = AcudaSpline[tid];
  __syncthreads();


  int Nc = clast - cfirst + 1;
  int Ne = elast - efirst + 1;
  int NBc = Nc/BS + ((Nc % BS) ? 1 : 0);
  int NBe = Ne/BS + ((Ne % BS) ? 1 : 0);

  T mysum = (T)0.0; 
  for (int bc=0; bc < NBc; bc++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*bc+i)*BS + tid < 3*Nc) 
  	rc[0][i*BS + tid] = C[3*cfirst + (3*bc+i)*BS + tid];
    __syncthreads();
    int ptcl1 = cfirst+bc*BS + tid;
    for (int be=0; be < NBe; be++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*be+i)*BS + tid < 3*Ne) 
	  re[0][i*BS + tid] = myR[3*efirst + (3*be+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (be+1)*BS < Ne ? BS : Ne-be*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = efirst + be*BS+j;
  	T dx, dy, dz;
  	dx = re[j][0] - rc[tid][0];
  	dy = re[j][1] - rc[tid][1];
  	dz = re[j][2] - rc[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
  	if ((ptcl1 < (Nc+cfirst) ) && (ptcl2 < (Ne+efirst)))
	  mysum += eval_1d_spline (dist, rMax, drInv, A, coefs);
      }
    }
    __syncthreads();
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s=BS>>1; s>0; s >>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  if (tid==0)
    sum[blockIdx.x] += shared_sum[0];

}

void
one_body_sum (float C[], float *R[], int cfirst, int clast, int efirst, int elast,
	      float spline_coefs[], int numCoefs, float rMax,  
	      float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_sum_kernel<float,BS><<<dimGrid,dimBlock>>>
    (C, R, cfirst, clast, efirst, elast, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_sum:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
one_body_sum (double C[], double *R[], int cfirst, int clast, int efirst, int elast,
	      double spline_coefs[], int numCoefs, double rMax,  
	      double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_sum_kernel<double,BS><<<dimGrid,dimBlock>>>
    (C, R, cfirst, clast, efirst, elast, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_sum2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



template<typename T, int BS>
__global__ void
one_body_ratio_kernel(T *C, T **R, int cfirst, int clast,
		      T *Rnew, int inew,
		      T *spline_coefs, int numCoefs, T rMax,  
		      T *lattice, T *latticeInv, T *sum)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  int tid = threadIdx.x;
  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T c[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();

  int Nc = clast - cfirst + 1;
  int NB = Nc/BS + ((Nc % BS) ? 1 : 0);

  __shared__ T shared_sum[BS];
  shared_sum[tid] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*Nc) 
  	c[0][n] = C[3*cfirst + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = cfirst+b*BS + tid;

    T dx, dy, dz;
    dx = myRnew[0] - c[tid][0];
    dy = myRnew[1] - c[tid][1];
    dz = myRnew[2] - c[tid][2];
    T dist = min_dist(dx, dy, dz, L, Linv);
    T delta = eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv);
    delta -= eval_1d_spline (dist, rMax, drInv, A, coefs);
    
    if (ptcl1 < (Nc+cfirst) )
      shared_sum[tid] += delta;

    __syncthreads();
  }
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  if (tid==0)
    sum[blockIdx.x] += shared_sum[0];
}




void
one_body_ratio (float C[], float *R[], int first, int last,
		float Rnew[], int inew,
		float spline_coefs[], int numCoefs, float rMax,  
		float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_ratio_kernel<float,BS><<<dimGrid,dimBlock>>>
    (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_ratio:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



void
one_body_ratio (double C[], double *R[], int first, int last,
		double Rnew[], int inew,
		double spline_coefs[], int numCoefs, double rMax,  
		double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  dim3 dimBlock(128);
  dim3 dimGrid(numWalkers);

  one_body_ratio_kernel<double,128><<<dimGrid,dimBlock>>>
    (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_ratio2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


template<typename T, int BS>
__global__ void
one_body_ratio_grad_kernel(T *C, T **R, int cfirst, int clast,
			   T *Rnew, int inew,
			   T *spline_coefs, int numCoefs, T rMax,  
			   T *lattice, T* latticeInv, bool zero,
			   T *ratio_grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  int tid = threadIdx.x;
  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T c[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  int index=0;
  __shared__ T images[27][3];
  if (tid < 3)
    for (T i=-1.0; i<=1.001; i+=1.0)
      for (T j=-1.0; j<=1.001; j+=1.0)
	for (T k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int Nc = clast - cfirst + 1;
  int NB = Nc/BS + ((Nc % BS) ? 1 : 0);

  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid] = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*Nc) 
  	c[0][n] = C[3*cfirst + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = cfirst+b*BS + tid;

    T dx, dy, dz, dist, delta, u, du, d2u;
    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv, images);
    delta =- eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRnew[0] - c[tid][0];
    dy = myRnew[1] - c[tid][1];
    dz = myRnew[2] - c[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv, images);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    delta += u;

    if (ptcl1 < (Nc+cfirst) ) {
      du /= dist;
      shared_sum[tid] += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) {
      shared_sum[tid] += shared_sum[tid+s];
      shared_grad[tid][0] += shared_grad[tid+s][0];
      shared_grad[tid][1] += shared_grad[tid+s][1];
      shared_grad[tid][2] += shared_grad[tid+s][2];
    }
    __syncthreads();
  }

  if (tid==0) {
    if (zero) {
      ratio_grad[4*blockIdx.x+0] = shared_sum[0];
      ratio_grad[4*blockIdx.x+1] = shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] = shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] = shared_grad[0][2];
    }
    else {
      ratio_grad[4*blockIdx.x+0] += shared_sum[0];
      ratio_grad[4*blockIdx.x+1] += shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] += shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] += shared_grad[0][2];
    }
  }
}




template<typename T, int BS>
__global__ void
one_body_ratio_grad_kernel_fast(T *C, T **R, int cfirst, int clast,
				T *Rnew, int inew,
				T *spline_coefs, int numCoefs, T rMax,  
				T *lattice, T *latticeInv, bool zero,
				T *ratio_grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  int tid = threadIdx.x;
  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
  }
  __syncthreads();

  __shared__ T coefs[MAX_COEFS];
  __shared__ T c[BS][3];
  __shared__ T L[3][3], Linv[3][3];

  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int Nc = clast - cfirst + 1;
  int NB = Nc/BS + ((Nc % BS) ? 1 : 0);

  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid] = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*Nc) 
  	c[0][n] = C[3*cfirst + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = cfirst+b*BS + tid;

    T dx, dy, dz, dist, delta, u, du, d2u;
    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    dist = min_dist_fast(dx, dy, dz, L, Linv);
    delta =- eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRnew[0] - c[tid][0];
    dy = myRnew[1] - c[tid][1];
    dz = myRnew[2] - c[tid][2];
    dist = min_dist_fast(dx, dy, dz, L, Linv);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    delta += u;

    if (ptcl1 < (Nc+cfirst) ) {
      du /= dist;
      shared_sum[tid] += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) {
      shared_sum[tid] += shared_sum[tid+s];
      shared_grad[tid][0] += shared_grad[tid+s][0];
      shared_grad[tid][1] += shared_grad[tid+s][1];
      shared_grad[tid][2] += shared_grad[tid+s][2];
    }
    __syncthreads();
  }

  if (tid==0) {
    if (zero) {
      ratio_grad[4*blockIdx.x+0] = shared_sum[0];
      ratio_grad[4*blockIdx.x+1] = shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] = shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] = shared_grad[0][2];
    }
    else {
      ratio_grad[4*blockIdx.x+0] += shared_sum[0];
      ratio_grad[4*blockIdx.x+1] += shared_grad[0][0];
      ratio_grad[4*blockIdx.x+2] += shared_grad[0][1];
      ratio_grad[4*blockIdx.x+3] += shared_grad[0][2];
    }
  }
}




void
one_body_ratio_grad (float C[], float *R[], int first, int last,
		     float Rnew[], int inew,
		     float spline_coefs[], int numCoefs, float rMax,  
		     float lattice[], float latticeInv[], bool zero,
		     float ratio_grad[], int numWalkers, 
		     bool use_fast_image)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  // if (use_fast_image)
  //   one_body_ratio_grad_kernel_fast<float,BS><<<dimGrid,dimBlock>>>
  //     (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
  //      lattice, latticeInv, zero, ratio_grad);
  // else
    one_body_ratio_grad_kernel<float,BS><<<dimGrid,dimBlock>>>
      (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
       lattice, latticeInv, zero, ratio_grad);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_ratio_grad:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
one_body_ratio_grad (double C[], double *R[], int first, int last,
		     double Rnew[], int inew,
		     double spline_coefs[], int numCoefs, double rMax,  
		     double lattice[], double latticeInv[], bool zero,
		     double ratio_grad[], int numWalkers, bool use_fast_image)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 32;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  
  // if (use_fast_image)
  //   one_body_ratio_grad_kernel_fast<double,BS><<<dimGrid,dimBlock>>>
  //     (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
  //      lattice, latticeInv, zero, ratio_grad);
  // else
    one_body_ratio_grad_kernel<double,BS><<<dimGrid,dimBlock>>>
      (C, R, first, last, Rnew, inew, spline_coefs, numCoefs, rMax, 
       lattice, latticeInv, zero, ratio_grad);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_ratio_grad2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}




template<typename T>
__global__ void
one_body_update_kernel (T **R, int N, int iat)
{
  __shared__ T* myR;
  if (threadIdx.x == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  
  if (threadIdx.x < 3)
    myR[3*iat + threadIdx.x] = myR[3*N + threadIdx.x];


}

void
one_body_update(float *R[], int N, int iat, int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  dim3 dimBlock(32);
  dim3 dimGrid(numWalkers);

  one_body_update_kernel<float><<<dimGrid, dimBlock>>> (R, N, iat);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_update:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}

void
one_body_update(double *R[], int N, int iat, int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  dim3 dimBlock(3);
  dim3 dimGrid(numWalkers);

  one_body_update_kernel<double><<<dimGrid, dimBlock>>> (R, N, iat);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_update:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}





template<typename T, int BS>
__global__ void
one_body_grad_lapl_kernel(T *C, T **R, int cfirst, int clast, 
			  int efirst, int elast,
			  T *spline_coefs, int numCoefs, T rMax,  
			  T *lattice, T* latticeInv, 
			  T *gradLapl, int row_stride)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;    

  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r[BS][3], c[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  int index=0;
  __shared__ T images[27][3];
  if (tid < 3)
    for (T i=-1.0; i<=1.001; i+=1.0)
      for (T j=-1.0; j<=1.001; j+=1.0)
	for (T k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();


  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  int Nc = clast - cfirst + 1;
  int Ne = elast - efirst + 1;
  int NBc = Nc/BS + ((Nc % BS) ? 1 : 0);
  int NBe = Ne/BS + ((Ne % BS) ? 1 : 0);

  __shared__ T sGradLapl[BS][4];
  for (int be=0; be < NBe; be++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*be+i)*BS + tid < 3*Ne) 
  	r[0][i*BS + tid] = myR[3*efirst + (3*be+i)*BS + tid];
    __syncthreads();
    int eptcl = efirst+be*BS + tid;
    int offset = blockIdx.x * row_stride + 4*be*BS + 4*efirst;
    sGradLapl[tid][0] = sGradLapl[tid][1] = 
      sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int bc=0; bc < NBc; bc++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*bc+i)*BS + tid < 3*Nc) 
	  c[0][i*BS + tid] = C[3*cfirst + (3*bc+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = ((bc+1)*BS < Nc) ? BS : Nc-bc*BS;
      for (int j=0; j<end; j++) {
  	int cptcl = cfirst + bc*BS+j;
  	T dx, dy, dz, u, du, d2u;
  	dx = r[tid][0] - c[j][0];
  	dy = r[tid][1] - c[j][1];
  	dz = r[tid][2] - c[j][2];
  	T dist = min_dist(dx, dy, dz, L, Linv, images);
	eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
  	if (cptcl < (Nc+cfirst)  && (eptcl < (Ne+efirst))) {
	  du /= dist;
	  sGradLapl[tid][0] -= du * dx;
	  sGradLapl[tid][1] -= du * dy;
	  sGradLapl[tid][2] -= du * dz;
	  sGradLapl[tid][3] -= d2u + 2.0*du;
	}
      }
      __syncthreads();
    }
    __syncthreads();
    for (int i=0; i<4; i++)
      if ((4*be+i)*BS + tid < 4*Ne)
	gradLapl[offset + i*BS +tid] += sGradLapl[0][i*BS+tid];
    __syncthreads();
  }
}


void
one_body_grad_lapl(float C[], float *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   float spline_coefs[], int numCoefs, float rMax,  
		   float lattice[], float latticeInv[], 
		   float gradLapl[], int row_stride, int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_grad_lapl_kernel<float,BS><<<dimGrid,dimBlock>>>
    (C, R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
     rMax, lattice, latticeInv,  gradLapl, row_stride);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_grad_lapl:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
one_body_grad_lapl(double C[], double *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   double spline_coefs[], int numCoefs, double rMax,  
		   double lattice[], double latticeInv[], 
		   double gradLapl[], int row_stride, int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_grad_lapl_kernel<double,BS><<<dimGrid,dimBlock>>>
    (C, R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
     rMax, lattice, latticeInv,  gradLapl, row_stride);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_grad_lapl:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


template<int BS>
__global__ void
one_body_NLratio_kernel(NLjobGPU<float> *jobs, float *C, int first, int last,
			float *spline_coefs, int numCoefs, float rMax, 
			float *lattice, float *latticeInv)
{
  const int MAX_RATIOS = 18;
  int tid = threadIdx.x;
  __shared__ NLjobGPU<float> myJob;
  __shared__ float myRnew[MAX_RATIOS][3], myRold[3];
  if (tid == 0) 
    myJob = jobs[blockIdx.x];
  __syncthreads();

  if (tid < 3 ) 
    myRold[tid] = myJob.R[3*myJob.Elec+tid];
  for (int i=0; i<3; i++) 
    if (i*BS + tid < 3*myJob.NumQuadPoints)
      myRnew[0][i*BS+tid] = myJob.QuadPoints[i*BS+tid];
  __syncthreads();

  float dr = rMax/(float)(numCoefs-3);
  float drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  __shared__ float coefs[MAX_COEFS];
  __shared__ float c[BS][3];
  __shared__ float L[3][3], Linv[3][3];
  
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  __syncthreads();

  int index=0;
  __shared__ float images[27][3];
  if (tid < 3)
    for (float i=-1.0; i<=1.001; i+=1.0)
      for (float j=-1.0; j<=1.001; j+=1.0)
	for (float k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();

  
  __shared__ float A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();
  
  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  
  __shared__ float shared_sum[MAX_RATIOS][BS+1];
  for (int iq=0; iq<myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (float)0.0;

  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	c[0][n] = C[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    float dx, dy, dz;
    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    float dist = min_dist_only(dx, dy, dz, L, Linv, images);
    float uOld = eval_1d_spline (dist, rMax, drInv, A, coefs);

    for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
      dx = myRnew[iq][0] - c[tid][0];
      dy = myRnew[iq][1] - c[tid][1];
      dz = myRnew[iq][2] - c[tid][2];
      dist = min_dist_only(dx, dy, dz, L, Linv, images);
      if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) 
      for (int iq=0; iq < myJob.NumQuadPoints; iq++)
	shared_sum[iq][tid] += shared_sum[iq][tid+s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}



template<int BS>
__global__ void
one_body_NLratio_kernel_fast(NLjobGPU<float> *jobs, float *C, int first, int last,
			     float *spline_coefs, int numCoefs, float rMax, 
			     float *lattice, float *latticeInv)
{
  const int MAX_RATIOS = 18;
  int tid = threadIdx.x;
  __shared__ NLjobGPU<float> myJob;
  __shared__ float myRnew[MAX_RATIOS][3], myRold[3];
  if (tid == 0) 
    myJob = jobs[blockIdx.x];
  __syncthreads();

  if (tid < 3 ) 
    myRold[tid] = myJob.R[3*myJob.Elec+tid];
  for (int i=0; i<3; i++) 
    if (i*BS + tid < 3*myJob.NumQuadPoints)
      myRnew[0][i*BS+tid] = myJob.QuadPoints[i*BS+tid];
  __syncthreads();

  float dr = rMax/(float)(numCoefs-3);
  float drInv = 1.0/dr;
    __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  __shared__ float coefs[MAX_COEFS];
  __shared__ float c[BS][3];
  __shared__ float L[3][3], Linv[3][3];
  
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ float A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();
  
  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  
  __shared__ float shared_sum[MAX_RATIOS][BS+1];
  for (int iq=0; iq<myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (float)0.0;

  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	c[0][n] = C[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    float dx, dy, dz;
    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    float dist = min_dist_fast(dx, dy, dz, L, Linv);
    float uOld = eval_1d_spline (dist, rMax, drInv, A, coefs);

    for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
      dx = myRnew[iq][0] - c[tid][0];
      dy = myRnew[iq][1] - c[tid][1];
      dz = myRnew[iq][2] - c[tid][2];
      dist = min_dist_fast(dx, dy, dz, L, Linv);
      if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) 
      for (int iq=0; iq < myJob.NumQuadPoints; iq++)
	shared_sum[iq][tid] += shared_sum[iq][tid+s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}






template<int BS>
__global__ void
one_body_NLratio_kernel(NLjobGPU<double> *jobs, double *C, int first, int last,
			double *spline_coefs, int numCoefs, double rMax, 
			double *lattice, double *latticeInv)
{
  const int MAX_RATIOS = 18;
  int tid = threadIdx.x;
  __shared__ NLjobGPU<double> myJob;
  __shared__ double myRnew[MAX_RATIOS][3], myRold[3];
  if (tid == 0) 
    myJob = jobs[blockIdx.x];
  __syncthreads();

  if (tid < 3 ) 
    myRold[tid] = myJob.R[3*myJob.Elec+tid];
  for (int i=0; i<3; i++) 
    if (i*BS + tid < 3*myJob.NumQuadPoints)
      myRnew[0][i*BS+tid] = myJob.QuadPoints[i*BS+tid];
  __syncthreads();

  double dr = rMax/(double)(numCoefs-3);
  double drInv = 1.0/dr;
  

  __shared__ double coefs[MAX_COEFS];
  __shared__ double c[BS][3];
  __shared__ double L[3][3], Linv[3][3];
  
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }

  __shared__ double images[27][3];
  int index=0;
  if (tid < 3)
    for (float i=-1.0; i<=1.001; i+=1.0)
      for (float j=-1.0; j<=1.001; j+=1.0)
	for (float k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();
  
  __shared__ double A[4][4];
  if (tid < 16) 
    A[(tid>>2)][tid&3] = AcudaSpline[tid];
  __syncthreads();
  
  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  
  __shared__ double shared_sum[MAX_RATIOS][BS+1];
  for (int iq=0; iq<myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (double)0.0;

  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++) {
      int n = i*BS + tid;
      if ((3*b+i)*BS + tid < 3*N) 
  	c[0][n] = C[3*first + (3*b+i)*BS + tid];
    }
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    double dx, dy, dz;
    dx = myRold[0] - c[tid][0];
    dy = myRold[1] - c[tid][1];
    dz = myRold[2] - c[tid][2];
    double dist = min_dist(dx, dy, dz, L, Linv, images);
    double uOld = eval_1d_spline (dist, rMax, drInv, A, coefs);

    for (int iq=0; iq<myJob.NumQuadPoints; iq++) {
      dx = myRnew[iq][0] - c[tid][0];
      dy = myRnew[iq][1] - c[tid][1];
      dz = myRnew[iq][2] - c[tid][2];
      dist = min_dist(dx, dy, dz, L, Linv, images);
      if (ptcl1 != myJob.Elec && (ptcl1 < (N+first)))
	shared_sum[iq][tid] += eval_1d_spline (dist, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  
  for (int s=(BS>>1); s>0; s>>=1) {
    if (tid < s) 
      for (int iq=0; iq < myJob.NumQuadPoints; iq++)
	shared_sum[iq][tid] += shared_sum[iq][tid+s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}





void
one_body_NLratios(NLjobGPU<float> jobs[], float C[], int first, int last,
		  float spline_coefs[], int numCoefs, float rMax, 
		  float lattice[], float latticeInv[], float sim_cell_radius,
		  int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;

  dim3 dimBlock(BS);

  while (numjobs > 65535) {
    dim3 dimGrid(65535);
    if (rMax <= sim_cell_radius)
      one_body_NLratio_kernel_fast<BS><<<dimGrid,dimBlock>>>
	(jobs, C, first, last, spline_coefs, numCoefs, rMax,
	 lattice, latticeInv);
    else
      one_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
	(jobs, C, first, last, spline_coefs, numCoefs, rMax,
	 lattice, latticeInv);
    numjobs -= 65535;
    jobs += 65535;
  }

  dim3 dimGrid(numjobs);
  if (rMax <= sim_cell_radius)
    one_body_NLratio_kernel_fast<BS><<<dimGrid,dimBlock>>>
      (jobs, C, first, last, spline_coefs, numCoefs, rMax,
       lattice, latticeInv);
  else
    one_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
      (jobs, C, first, last, spline_coefs, numCoefs, rMax,
       lattice, latticeInv);
  
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_NLratios:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}


void
one_body_NLratios(NLjobGPU<double> jobs[], double C[], int first, int last,
		 double spline_coefs[], int numCoefs, double rMax, 
		 double lattice[], double latticeInv[], int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS=32;

  dim3 dimBlock(BS);
  int blockx = numjobs % 65535;
  int blocky = numjobs / 65535 + 1;
  dim3 dimGrid(blockx, blocky);

  one_body_NLratio_kernel<BS><<<dimGrid,dimBlock>>>
    (jobs, C, first, last, spline_coefs, numCoefs, rMax,
     lattice, latticeInv);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_NLratios2:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



template<typename T, int BS>
__global__ void
one_body_grad_kernel(T **R, int iat, T *C, int first, int last,
		     T *spline_coefs, int numCoefs, T rMax,  
		     T *lattice, T *latticeInv, bool zeroOut, T* grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  
  

  int tid = threadIdx.x;
  __shared__ T *myR, r[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r[tid] = myR[3*iat+tid];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T c[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int index=0;
  __shared__ T images[27][3];
  if (tid < 3)
    for (T i=-1.0; i<=1.001; i+=1.0)
      for (T j=-1.0; j<=1.001; j+=1.0)
	for (T k=-1.0; k<=1.001; k+=1.0) {
	  images[index][tid] = 
	    i*L[0][tid] + j*L[1][tid] + k*L[2][tid];
	    index++;
	}
  __syncthreads();

  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0]   = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b+i)*BS + tid < 3*N) 
  	c[0][i*BS + tid] = C[3*first + (3*b+i)*BS + tid];
    __syncthreads();
    int ptcl1 = first+b*BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx = r[0] - c[tid][0];
    dy = r[1] - c[tid][1];
    dz = r[2] - c[tid][2];
    T dist = min_dist(dx, dy, dz, L, Linv, images);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 < (N+first)) {
      du /= dist;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s=BS>>1; s>0; s>>=1) {
    if (tid < s) {
      sGrad[tid][0] += sGrad[tid+s][0];
      sGrad[tid][1] += sGrad[tid+s][1];
      sGrad[tid][2] += sGrad[tid+s][2];
    }
    __syncthreads();
  }
  if (tid < 3) {
    if (zeroOut) 
      grad[3*blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3*blockIdx.x + tid] += sGrad[0][tid];
  }
}


template<typename T, int BS>
__global__ void
one_body_grad_kernel_fast(T **R, int iat, T *C, int first, int last,
			  T *spline_coefs, int numCoefs, T rMax,  
			  T *lattice, T *latticeInv, bool zeroOut, T *grad)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  
  int tid = threadIdx.x;
  __shared__ T *myR, r[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r[tid] = myR[3*iat+tid];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T c[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();

  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0]   = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b+i)*BS + tid < 3*N) 
  	c[0][i*BS + tid] = C[3*first + (3*b+i)*BS + tid];
    __syncthreads();
    int ptcl1 = first+b*BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx = r[0] - c[tid][0];
    dy = r[1] - c[tid][1];
    dz = r[2] - c[tid][2];
    T dist = min_dist_fast(dx, dy, dz, L, Linv);
    eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 < (N+first)) {
      du /= dist;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s=BS>>1; s>0; s>>=1) {
    if (tid < s) {
      sGrad[tid][0] += sGrad[tid+s][0];
      sGrad[tid][1] += sGrad[tid+s][1];
      sGrad[tid][2] += sGrad[tid+s][2];
    }
    __syncthreads();
  }
  if (tid < 3) {
    if (zeroOut) 
      grad[3*blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3*blockIdx.x + tid] += sGrad[0][tid];
  }
}




void
one_body_gradient (float *Rlist[], int iat, float C[], int first, int last,
		   float spline_coefs[], int num_coefs, float rMax,
		   float L[], float Linv[], float sim_cell_radius,
		   bool zeroSum, float grad[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  // if (sim_cell_radius >= rMax)
  //   one_body_grad_kernel_fast<float,BS><<<dimGrid,dimBlock>>>
  //     (Rlist, iat, C, first, last, spline_coefs, num_coefs, rMax,
  //      L, Linv, zeroSum, grad);
  // else
    one_body_grad_kernel<float,BS><<<dimGrid,dimBlock>>>
      (Rlist, iat, C, first, last, spline_coefs, num_coefs, rMax,
       L, Linv, zeroSum, grad);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_gradient:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }

}
		   

void
one_body_gradient (double *Rlist[], int iat, double C[], int first, int last,
		   double spline_coefs[], int num_coefs, double rMax,
		   double L[], double Linv[], bool zeroSum,
		   double grad[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_grad_kernel<double,BS><<<dimGrid,dimBlock>>>
    (Rlist, iat, C, first, last, spline_coefs, num_coefs, rMax,
     L, Linv, zeroSum, grad);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_gradient1:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



template<typename T, int BS>
__global__ void
one_body_derivs_kernel(T* C, T **R, T **gradLogPsi,
		       int cfirst, int clast, 
		       int efirst, int elast,
		       int numCoefs, T rMax,  
		       T *lattice, T *latticeInv, 
		       T **derivs)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999;  

  
  int tid = threadIdx.x;
  __shared__ T *myR, *myGrad, *myDerivs;
  if (tid == 0) {
    myR      =          R[blockIdx.x];
    myGrad   = gradLogPsi[blockIdx.x];
    myDerivs =     derivs[blockIdx.x];
  }
    
  __shared__ T sderivs[MAX_COEFS][2];
  __shared__ T r[BS][3], c[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  sderivs[tid][0] = T();
  sderivs[tid][1] = T();

  int Nc = clast - cfirst + 1;
  int Ne = elast - efirst + 1;
  int NBc = (Nc+BS-1)/BS;
  int NBe = (Ne+BS-1)/BS;

  __shared__ T sGrad[BS][3];
  for (int be=0; be < NBe; be++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*be+i)*BS + tid < 3*Ne) {
	int outoff = i*BS+tid;
	int inoff  = outoff + 3*efirst + 3*be*BS;
  	r[0][outoff]    =     myR[inoff];
	sGrad[0][outoff] = myGrad[inoff];
      }
    __syncthreads();
    int eptcl = efirst+be*BS + tid;
    for (int bc=0; bc < NBc; bc++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*bc+i)*BS + tid < 3*Nc) 
	  c[0][i*BS + tid] = C[3*cfirst + (3*bc+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = min(BS, Nc-bc*BS);
      for (int j=0; j<end; j++) {
  	T dx, dy, dz;
  	dx = c[j][0] - r[tid][0];
  	dy = c[j][1] - r[tid][1];
  	dz = c[j][2] - r[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
	T distInv = 1.0f/dist;

	T s = dist * drInv;
	T sf = floorf (s);
	int index = (int)sf;
	T t = s - sf;
	T t2 = t*t;
	T t3 = t*t2;
	
	T v0 = (A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]);
	T v1 = (A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]);
	T v2 = (A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]);
	T v3 = (A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]);

	for (int id=0; id<BS; id++)
	  if (tid == id && eptcl <= elast && (dist < rMax)) {
	    sderivs[index+0][0] += v0;
	    sderivs[index+1][0] += v1;
	    sderivs[index+2][0] += v2;
	    sderivs[index+3][0] += v3;
	  }
	T prefact = (dx*sGrad[tid][0] + dy*sGrad[tid][1] + dz*sGrad[tid][2])*distInv;
	T du0 = drInv * (A[4][0]*t3 + A[4][1]*t2 + A[4][2]*t + A[4][3]);
	T du1 = drInv * (A[5][0]*t3 + A[5][1]*t2 + A[5][2]*t + A[5][3]);
	T du2 = drInv * (A[6][0]*t3 + A[6][1]*t2 + A[6][2]*t + A[6][3]);
	T du3 = drInv * (A[7][0]*t3 + A[7][1]*t2 + A[7][2]*t + A[7][3]);
	// This is the dot (gradu, grad_log_psi) term.
	v0 = 2.0f* prefact * du0;
	v1 = 2.0f* prefact * du1;
	v2 = 2.0f* prefact * du2;
	v3 = 2.0f* prefact * du3;
	// This is the lapl u term
	v0 -= drInv*drInv*(A[ 8][0]*t3 + A[ 8][1]*t2 + A[ 8][2]*t + A[ 8][3]) + 2.0f*du0*distInv;
	v1 -= drInv*drInv*(A[ 9][0]*t3 + A[ 9][1]*t2 + A[ 9][2]*t + A[ 9][3]) + 2.0f*du1*distInv;
	v2 -= drInv*drInv*(A[10][0]*t3 + A[10][1]*t2 + A[10][2]*t + A[10][3]) + 2.0f*du2*distInv;
	v3 -= drInv*drInv*(A[11][0]*t3 + A[11][1]*t2 + A[11][2]*t + A[11][3]) + 2.0f*du3*distInv;
	for (int id=0; id<BS; id++)
	  if (tid == id && eptcl <= elast && (dist < rMax)) {
	    sderivs[index+0][1] += v0;
	    sderivs[index+1][1] += v1;
	    sderivs[index+2][1] += v2;
	    sderivs[index+3][1] += v3;
	  }
      }
      __syncthreads();
    }
  }
  
  sderivs[tid][1] *= 0.5f;

  if (tid < 2*numCoefs) 
    myDerivs[tid] = -sderivs[0][tid];
  if (tid+BS < 2*numCoefs)
    myDerivs[tid+BS] = -sderivs[0][tid+BS];
  
}


void
one_body_derivs(float C[], float *R[], float *gradLogPsi[], 
		int cfirst, int clast, 
		int efirst, int elast,
		int numCoefs, float rMax,  
		float lattice[], float latticeInv[], float sim_cell_radius,
		float *derivs[], int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  if (sim_cell_radius >= rMax) 
    one_body_derivs_kernel<float,BS><<<dimGrid,dimBlock>>>
      (C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, 
       rMax, lattice, latticeInv, derivs);
  else
    one_body_derivs_kernel<float,BS><<<dimGrid,dimBlock>>>
      (C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, 
       rMax, lattice, latticeInv, derivs);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_derivs:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}



void
one_body_derivs(double C[], double *R[], double *gradLogPsi[], 
		int cfirst, int clast, 
		int efirst, int elast,
		int numCoefs, double rMax,  
		double lattice[], double latticeInv[], double sim_cell_radius,
		double *derivs[], int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  if (sim_cell_radius >= rMax) 
    one_body_derivs_kernel<double,BS><<<dimGrid,dimBlock>>>
      (C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, 
       rMax, lattice, latticeInv, derivs);
  else
    one_body_derivs_kernel<double,BS><<<dimGrid,dimBlock>>>
      (C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, 
       rMax, lattice, latticeInv, derivs);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf (stderr, "CUDA error in one_body_derivs:\n  %s\n",
	     hipGetErrorString(err));
    abort();
  }
}




void test()
{
  dim3 dimBlock(32);
  dim3 dimGrid(1000);

  float *R[1000];
  float L[9], Linv[9];
  float spline_coefs[10];
  float dr = 0.1;
  float sum[1000];

  two_body_sum_kernel<float,32><<<dimGrid,dimBlock>>>(R, 0, 100, 0, 100, spline_coefs, 10, dr,
						      L, Linv, sum);



}
