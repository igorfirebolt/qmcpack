#define MAX_SPLINES 100

#include <hip/hip_runtime.h>
#include <stdio.h>

bool AisInitialized = false;


// void
// createCudaSplines (float rmax, int N,
// 		   float f[], float df[], float d2f[],
// 		   int &fSpline, int &dfSpline, int &d2fSpline)
// {
//   cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<float>();
//   cudaArray *fArray, *dfArray, *d2fArray;
//   cudaMallocArray(  &fArray, &channelDesc, N);
//   cudaMallocArray( &dfArray, &channelDesc, N);
//   cudaMallocArray(&d2fArray, &channelDesc, N);
  
//   cudaMemcpyToArray(fArray,  N,1,  f,N*sizeof(float),cudaMemcpyHostToDevice);
//   cudaMemcpyToArray(dfArray, N,1, df,N*sizeof(float),cudaMemcpyHostToDevice);
//   cudaMemcpyToArray(d2fArray,N,1,d2f,N*sizeof(float),cudaMemcpyHostToDevice);


//   cudaBindTextureToArray(texSplines[fSpline=curTex++], fArray);
//   cudaBindTextureToArray(texSplines[dfSpline=curTex++], dfArray);
//   cudaBindTextureToArray(texSplines[d2fSpline=curTex++], d2fArray);
// }


template<typename T>
__device__
T min_dist (T& x, T& y, T& z, 
	    T L[3][3], T Linv[3][3])
{
//   T u0 = Linv[0][0]*x + Linv[0][1]*y + Linv[0][2]*z;  
//   T u1 = Linv[1][0]*x + Linv[1][1]*y + Linv[1][2]*z;
//   T u2 = Linv[2][0]*x + Linv[2][1]*y + Linv[2][2]*z;

//   u0 -= rintf(u0);
//   u1 -= rintf(u1);
//   u2 -= rintf(u2);

//   x = L[0][0]*u0 + L[0][1]*u1 + L[0][2]*u2;
//   y = L[1][0]*u0 + L[1][1]*u1 + L[1][2]*u2;
//   z = L[2][0]*u0 + L[2][1]*u1 + L[2][2]*u2;

  T u0 = Linv[0][0]*x; u0 -= rintf(u0); x = L[0][0]*u0;
  T u1 = Linv[1][1]*y; u1 -= rintf(u1); y = L[1][1]*u1;
  T u2 = Linv[2][2]*z; u2 -= rintf(u2); z = L[2][2]*u2;

  return sqrtf(x*x + y*y + z*z);

//   T d2min = x*x + y*y + z*z;
//   for (T i=-1.0f; i<=1.001; i+=1.0f)
//     for (T j=-1.0f; j<=1.001; j+=1.0f)
//       for (T k=-1.0f; k<=1.001; k+=1.0f) {
// 	T xnew = L[0][0]*(u0+i) + L[0][1]*(u1+j) + L[0][2]*(u2+k);
// 	T ynew = L[1][0]*(u0+i) + L[1][1]*(u1+j) + L[1][2]*(u2+k);
// 	T znew = L[2][0]*(u0+i) + L[2][1]*(u1+j) + L[2][2]*(u2+k);
	
// 	T d2 = xnew*xnew + ynew*ynew + znew*znew;
// 	d2min = min (d2, d2min);
// 	if (d2 < d2min) {
// 	  d2min = d2;
// 	  x = xnew;
// 	  y = ynew;
// 	  z = znew;
// 	}
//       }
//   return sqrt(d2min);
}

__constant__ float AcudaSpline[48];
__constant__ double AcudaSpline_double[48];

void
cuda_spline_init()
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  double A_d[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline_double), A_d, 48*sizeof(double), 0, hipMemcpyHostToDevice);

  AisInitialized = true;
}


template<typename T>
__device__ T 
eval_1d_spline(T dist, T rmax, T drInv, T A[4][4], T coefs[])
{
  if (dist > rmax)  return (T)0.0;

  T s = dist * drInv;
  T sf = floorf (s);
  int index = (int)sf;
  T t = s - sf;

//   return (coefs[index+0]*(AcudaSpline[ 0]*t*t*t + AcudaSpline[ 1]*t*t + AcudaSpline[ 2]*t + AcudaSpline[ 3]) +
//   	  coefs[index+1]*(AcudaSpline[ 4]*t*t*t + AcudaSpline[ 5]*t*t + AcudaSpline[ 6]*t + AcudaSpline[ 7]) +
//   	  coefs[index+2]*(AcudaSpline[ 8]*t*t*t + AcudaSpline[ 9]*t*t + AcudaSpline[10]*t + AcudaSpline[11]) +
//   	  coefs[index+3]*(AcudaSpline[12]*t*t*t + AcudaSpline[13]*t*t + AcudaSpline[14]*t + AcudaSpline[15]));


  return (coefs[index+0]*(A[0][0]*t*t*t + A[0][1]*t*t + A[0][2]*t + A[0][3]) +
  	  coefs[index+1]*(A[1][0]*t*t*t + A[1][1]*t*t + A[1][2]*t + A[1][3]) +
  	  coefs[index+2]*(A[2][0]*t*t*t + A[2][1]*t*t + A[2][2]*t + A[2][3]) +
  	  coefs[index+3]*(A[3][0]*t*t*t + A[3][1]*t*t + A[3][2]*t + A[3][3]));
}


template<typename T>
__device__ void 
eval_1d_spline_vgl(T dist, T rmax, T drInv, T A[12][4], T coefs[],
		   T& u, T& du, T& d2u)
{
  if (dist > rmax) {
    u = du = d2u = (T)0.0;
    return;
  }

  T s = dist * drInv;
  T sf = floorf (s);
  int index = (int)sf;
  T t = s - sf;

  u = (coefs[index+0]*(A[0][0]*t*t*t + A[0][1]*t*t + A[0][2]*t + A[0][3]) +
       coefs[index+1]*(A[1][0]*t*t*t + A[1][1]*t*t + A[1][2]*t + A[1][3]) +
       coefs[index+2]*(A[2][0]*t*t*t + A[2][1]*t*t + A[2][2]*t + A[2][3]) +
       coefs[index+3]*(A[3][0]*t*t*t + A[3][1]*t*t + A[3][2]*t + A[3][3]));

 du = drInv *    
   (coefs[index+0]*(A[4][0]*t*t*t + A[4][1]*t*t + A[4][2]*t + A[4][3]) +
    coefs[index+1]*(A[5][0]*t*t*t + A[5][1]*t*t + A[5][2]*t + A[5][3]) +
    coefs[index+2]*(A[6][0]*t*t*t + A[6][1]*t*t + A[6][2]*t + A[6][3]) +
    coefs[index+3]*(A[7][0]*t*t*t + A[7][1]*t*t + A[7][2]*t + A[7][3]));
 
 d2u = drInv*drInv * 
   (coefs[index+0]*(A[ 8][0]*t*t*t + A[ 8][1]*t*t + A[ 8][2]*t + A[ 8][3]) +
    coefs[index+1]*(A[ 9][0]*t*t*t + A[ 9][1]*t*t + A[ 9][2]*t + A[ 9][3]) +
    coefs[index+2]*(A[10][0]*t*t*t + A[10][1]*t*t + A[10][2]*t + A[10][3]) +
    coefs[index+3]*(A[11][0]*t*t*t + A[11][1]*t*t + A[11][2]*t + A[11][3]));
}



#define MAX_COEFS 32
template<typename T, int BS >
__global__ void
two_body_sum_kernel(T *R[], int e1_first, int e1_last, 
		    int e2_first, int e2_last,
		    T spline_coefs[], int numCoefs, T rMax,  
		    T lattice[], T latticeInv[], T sum[])
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;

  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[4][4];
  if (tid < 16)
    A[tid>>2][tid&3] = AcudaSpline_double[tid];
  __syncthreads();


  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  T mysum = (T)0.0; 
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) 
  	r1[0][i*BS + tid] = myR[3*e1_first + (3*b1+i)*BS + tid];
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
  	if (ptcl1 != ptcl2 && (ptcl1 < (N1+e1_first) ) && (ptcl2 < (N2+e2_first)))
	  mysum += eval_1d_spline (dist, rMax, drInv, A, coefs);
      }

    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s=BS>>1; s>0; s >>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  T factor = (e1_first == e2_first) ? 0.5 : 1.0;

  if (tid==0)
    sum[blockIdx.x] += factor*shared_sum[0];

}

void
two_body_sum (float *R[], int e1_first, int e1_last, int e2_first, int e2_last,
	      float spline_coefs[], int numCoefs, float rMax,  
	      float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_sum_kernel<float,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
}


void
two_body_sum (double *R[], int e1_first, int e1_last, int e2_first, int e2_last,
	      double spline_coefs[], int numCoefs, double rMax,  
	      double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_sum_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, 
     spline_coefs, numCoefs, rMax, lattice, latticeInv, sum);
}




template<typename T, int BS>
__global__ void
two_body_ratio_kernel(T *R[], int first, int last, int Ntotal,
		      T  Rnew[], int inew,
		      T spline_coefs[], int numCoefs, T rMax,  
		      T lattice[], T latticeInv[], T sum[])
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;

  int tid = threadIdx.x;
  __shared__ T *myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0) 
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3 ) {
    myRnew[tid] = Rnew[3*blockIdx.x+tid];
    myRold[tid] = myR[3*inew+tid];
    // Store RNew into R, so that we can copy it if we wish to accept the move
    myR[3*Ntotal+tid] = myRnew[tid];
  }

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  
  __shared__ T A[4][4];
  if (tid < 16) 
    A[tid>>2][tid&3] = AcudaSpline_double[tid];
  __syncthreads();

  int N = last - first + 1;
  int NB = N/BS + ((N % BS) ? 1 : 0);

  double mysum = (T)0.0; 
  for (int b=0; b < NB; b++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b+i)*BS + tid < 3*N) 
  	r1[0][i*BS + tid] = myR[3*first + (3*b+i)*BS + tid];
    __syncthreads();
    int ptcl1 = first+b*BS + tid;

    T dx, dy, dz;
    dx = myRnew[0] - r1[tid][0];
    dy = myRnew[1] - r1[tid][1];
    dz = myRnew[2] - r1[tid][2];
    T dist = min_dist(dx, dy, dz, L, Linv);
    T delta = eval_1d_spline (dist, rMax, drInv, A, coefs);

    dx = myRold[0] - r1[tid][0];
    dy = myRold[1] - r1[tid][1];
    dz = myRold[2] - r1[tid][2];
    dist = min_dist(dx, dy, dz, L, Linv);
    delta -= eval_1d_spline (dist, rMax, drInv, A, coefs);
    
    if (ptcl1 != inew && (ptcl1 < (N+first) ))
      mysum += delta;
  }
  
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s=BS>>1; s>0; s >>=1) {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid+s];
    __syncthreads();
  }

  //  float factor = (first <= inew && inew <= last) ? 0.5 : 1.0;
  if (tid==0)
    sum[blockIdx.x] += /* factor * */ shared_sum[0];
}




void
two_body_ratio (float *R[], int first, int last, int N,
		float Rnew[], int inew,
		float spline_coefs[], int numCoefs, float rMax,  
		float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  const int BS = 128;

  //  int numBlocks = numWalkers / BS + ((numWalkers %BS) ? 1 :
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_ratio_kernel<float,BS><<<dimGrid,dimBlock>>>
    (R, first, last, N, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
}



void
two_body_ratio (double *R[], int first, int last, int N,
		double Rnew[], int inew,
		double spline_coefs[], int numCoefs, double rMax,  
		double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();

  dim3 dimBlock(128);
  dim3 dimGrid(numWalkers);

  two_body_ratio_kernel<double,128><<<dimGrid,dimBlock>>>
    (R, first, last, N, Rnew, inew, spline_coefs, numCoefs, rMax, 
     lattice, latticeInv, sum);
}


template<typename T>
__global__ void
two_body_update_kernel (T *R[], int N, int iat)
{
  __shared__ T* myR;
  if (threadIdx.x == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  
  if (threadIdx.x < 3)
    myR[3*iat + threadIdx.x] = myR[3*N + threadIdx.x];
}

void
two_body_update(float *R[], int N, int iat, int numWalkers)
{
  dim3 dimBlock(3);
  dim3 dimGrid(numWalkers);

  two_body_update_kernel<float><<<dimGrid, dimBlock>>> (R, N, iat);
}

void
two_body_update(double *R[], int N, int iat, int numWalkers)
{
  dim3 dimBlock(3);
  dim3 dimGrid(numWalkers);

  two_body_update_kernel<double><<<dimGrid, dimBlock>>> (R, N, iat);
}





#define MAX_COEFS 32

template<typename T, int BS>
__global__ void
two_body_grad_lapl_kernel(T *R[], int e1_first, int e1_last, 
			  int e2_first, int e2_last,
			  T spline_coefs[], int numCoefs, T rMax,  
			  T lattice[], T latticeInv[], 
			  T gradLapl[], int row_stride)
{
  T dr = rMax/(T)(numCoefs-3);
  T drInv = 1.0/dr;
  
  T factor = (e1_first == e2_first) ? 0.5 : 1.0;


  int tid = threadIdx.x;
  __shared__ T *myR;
  if (tid == 0) 
    myR = R[blockIdx.x];

  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9) {
    L[0][tid] = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  

  __shared__ T A[12][4];
  if (tid < 16) {
    A[0+(tid>>2)][tid&3] = AcudaSpline[tid+0];
    A[4+(tid>>2)][tid&3] = AcudaSpline[tid+16];
    A[8+(tid>>2)][tid&3] = AcudaSpline[tid+32];
  }
  __syncthreads();


  int N1 = e1_last - e1_first + 1;
  int N2 = e2_last - e2_first + 1;
  int NB1 = N1/BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2/BS + ((N2 % BS) ? 1 : 0);

  __shared__ T sGradLapl[BS][4];
  for (int b1=0; b1 < NB1; b1++) {
    // Load block of positions from global memory
    for (int i=0; i<3; i++)
      if ((3*b1+i)*BS + tid < 3*N1) 
  	r1[0][i*BS + tid] = myR[3*e1_first + (3*b1+i)*BS + tid];
    __syncthreads();
    int ptcl1 = e1_first+b1*BS + tid;
    int offset = blockIdx.x * row_stride + 4*b1*BS;
    sGradLapl[tid][0] = sGradLapl[tid][1] = 
      sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int b2=0; b2 < NB2; b2++) {
      // Load block of positions from global memory
      for (int i=0; i<3; i++)
  	if ((3*b2+i)*BS + tid < 3*N2) 
	  r2[0][i*BS + tid] = myR[3*e2_first + (3*b2+i)*BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2+1)*BS < N2 ? BS : N2-b2*BS;
      for (int j=0; j<end; j++) {
  	int ptcl2 = e2_first + b2*BS+j;
  	T dx, dy, dz, u, du, d2u;
  	dx = r2[j][0] - r1[tid][0];
  	dy = r2[j][1] - r1[tid][1];
  	dz = r2[j][2] - r1[tid][2];
  	T dist = min_dist(dx, dy, dz, L, Linv);
	eval_1d_spline_vgl (dist, rMax, drInv, A, coefs, u, du, d2u);
  	if (ptcl1 != ptcl2 && (ptcl1 < (N1+e1_first) ) && (ptcl2 < (N2+e2_first))) {
	  du /= dist;
	  sGradLapl[tid][0] -= du * dx;
	  sGradLapl[tid][1] -= du * dy;
	  sGradLapl[tid][2] -= du * dz;
	  sGradLapl[tid][3] -= d2u + 2.0*du;
	}
      }
    }
    __syncthreads();
    for (int i=0; i<4; i++)
      if ((4*b1+i)*BS + tid < 4*N1)
	gradLapl[offset + i*BS +tid] += /*factor**/sGradLapl[0][i*BS+tid];
    __syncthreads();
  }
}


void
two_body_grad_lapl(float *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   float spline_coefs[], int numCoefs, float rMax,  
		   float lattice[], float latticeInv[], 
		   float gradLapl[], int row_stride, int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_grad_lapl_kernel<float,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
     rMax, lattice, latticeInv,  gradLapl, row_stride);
}


void
two_body_grad_lapl(double *R[], int e1_first, int e1_last, 
		   int e2_first, int e2_last,
		   double spline_coefs[], int numCoefs, double rMax,  
		   double lattice[], double latticeInv[], 
		   double gradLapl[], int row_stride, int numWalkers)
{
  const int BS=32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_grad_lapl_kernel<double,BS><<<dimGrid,dimBlock>>>
    (R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, 
     rMax, lattice, latticeInv,  gradLapl, row_stride);
}






void test()
{
  dim3 dimBlock(32);
  dim3 dimGrid(1000);

  float *R[1000];
  float L[9], Linv[9];
  float spline_coefs[10];
  float dr = 0.1;
  float sum[1000];

  two_body_sum_kernel<float,32><<<dimGrid,dimBlock>>>(R, 0, 100, 0, 100, spline_coefs, 10, dr,
						      L, Linv, sum);



}
